#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include<hiprand/hiprand_kernel.h>
#include "MonteGpu.hh"
#define min(a,b) a<b?a:b
#define max(a,b) a>b?a:b

// cudamalloc functions

double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

__device__ int MBSortT(double RandomNum,double iEnergyBin,double * CF,double ISIZE,double NumPoints){
  int ISTEP,INCR,I;
  ISTEP = ISIZE;
  INCR = 0;
  for(int K = 0;K<12;++K){
      I = INCR;
      if(ISTEP==2){
        if(I==0){
          return I;
        }
        return I - 1;
      }
      I = INCR + ISTEP/2;
      if (I<= NumPoints){
        if(CF[(int)iEnergyBin*290+I]<RandomNum){
          INCR +=ISTEP;
        }
      }
      ISTEP = ISTEP/2;
  }
  if(I==0){
    return I;
  }
  return I - 1;
}


__device__ extern void GetCollisions(double *ElectronEnergyStep, double* MaxCollisionFreqTotal,double* BP,double*  F1,
  double*  F2,double* Sqrt2M,double* TwoM,double* TwoPi,double* MaxCollisionFreq,double * VTMB,double * TimeSum,
  double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
  double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * T,double * AP,double * TotalCollisionFrequency,int i,hiprandState* globalState){

  // function start
  int MaxBoltzNumsUsed = 0;
  hiprand_uniform(globalState);
  //R = hiprand_uniform( &state );
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = hiprand_uniform(globalState);
    R2 = hiprand_uniform(globalState);
    RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*TwoPi)));
    RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*TwoPi)));
  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = hiprand_uniform(globalState);
    T[i] = -1 * log(RandomNum)/(*MaxCollisionFreqTotal)+TDash;
    TDash = T[i];
    AP[i] = DirCosineZ1[i]*(*F2)*sqrt(EBefore[i]);
    EAfter = EBefore[i]+(AP[i]+(*BP)*T[i])*T[i];
    VelocityRatio = sqrt(EBefore[i]/EAfter);
    DCosineZ2 = DirCosineZ1[i] * VelocityRatio + T[i] * (*F2) / (2.0 * sqrt(EAfter));
    DCosineX2 = DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;

    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){
        R1 = hiprand_uniform(globalState);
        R2 = hiprand_uniform(globalState);
        RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*TwoPi)));
      }
      MaxBoltzNumsUsed = 1;
    }
    GasVelX[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelY[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelZ[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    VelocityX[i] = DCosineX2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityY[i] = DCosineY2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityZ[i] = DCosineZ2 * (*Sqrt2M) * sqrt(EAfter);
    COMEnergy[i] = (pow((VelocityX[i] - GasVelX[i]), 2) + pow((VelocityY[i] - GasVelY[i]), 2) + pow(
        (VelocityZ[i] - GasVelZ[i]),
        2)) / (*TwoM);
    iEnergyBins[i] = COMEnergy[i] / (*ElectronEnergyStep);
    iEnergyBins[i] = min(iEnergyBins[i], 3999);
    RandomNum = hiprand_uniform(globalState);

    TEST = TotalCollisionFrequency[(int)iEnergyBins[i]] / (*MaxCollisionFreq);
    if (RandomNum < TEST){
      TimeSum[i]+=T[i];
      return;
    }
  }
}

__device__ void ProcessCollisions(double *COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * AP, double * X,double * Y,double * Z, double *DirCosineX1,double *DirCosineY1,double *DirCosineZ1,double * iEnergyBin,double * CF,double * RGAS,double * EnergyLevels,
  double * INDEX, double * ANGCT, double * SCA, double * IPN, double * AngleFromZ, double * TwoPi, double * EBefore, double * Sqrt2M,
  double * TwoM,double *T,double * BP,double * F1,double * ISIZE,double * NumPoints,int i,hiprandState* globalState )
  {
    int I;
    double VelocityInCOM,DXCOM,DYCOM,DZCOM,T2,A,B,VelocityBefore,RandomNum;
    double S1,S2,EI,EXTRA,RandomNum2,CosTheta,EpsilonOkhr,Theta,Phi,SinPhi,CosPhi;
    double ARG1,D,U,Q,CosZAngle,SinZAngle,ARGZ,CONST12;
    double VXLab,VYLab,VZLab;
    VelocityInCOM  =  ((*Sqrt2M) * sqrt(COMEnergy[i]));

    DXCOM = (VelocityX[i] - GasVelX[i]) / VelocityInCOM;
    DYCOM = (VelocityY[i] - GasVelY[i]) / VelocityInCOM;
    DZCOM = (VelocityZ[i] - GasVelZ[i]) / VelocityInCOM;

    T2 = T[i]*T[i];
    A = AP[i]*T[i];
    B = (*BP) * T2;

    VelocityBefore = (*Sqrt2M) * sqrt(EBefore[i]);

    A = T[i] * VelocityBefore;
    X[i] += DirCosineX1[i] * A;
    Y[i] += DirCosineY1[i] * A;
    Z[i] += DirCosineZ1[i] * A + T2 * (*F1);
    RandomNum = hiprand_uniform(globalState);

    I = MBSortT(RandomNum,iEnergyBin[i], CF,(*ISIZE),(*NumPoints));
    while(CF[(int)iEnergyBin[i]*290+I]<RandomNum) I+=1;


    S1 = RGAS[I];
    EI = EnergyLevels[I];

    if(IPN[I]>0){
      RandomNum = hiprand_uniform(globalState);
      EXTRA = RandomNum * (COMEnergy[i]-EI);
      EI = EXTRA + EI;
    }

    if(COMEnergy[i]<EI){
      EI = COMEnergy[i]-0.0001;
    }

    S2 = (S1*S1)/(S1 - 1.0);
    RandomNum = hiprand_uniform(globalState);

    if(INDEX[I] == 1){
      RandomNum2 = hiprand_uniform(globalState);
      CosTheta = 1.0-RandomNum*ANGCT[(int)iEnergyBin[i]*290 + I];
      if(RandomNum2>SCA[(int)iEnergyBin[i]*290 + I]){
        CosTheta = -1.0 * CosTheta;
      }
    }else if(INDEX[I]==2){
      EpsilonOkhr = SCA[(int)iEnergyBin[i]*290 + I];
      CosTheta = 1.0 - (2.0 * RandomNum * (1.0 - EpsilonOkhr) / (1.0 + EpsilonOkhr * (1.0 - 2.0 * RandomNum)));
    }else{
      CosTheta = 1.0 - 2.0*RandomNum;
    }

    Theta = acos(CosTheta);
    RandomNum = hiprand_uniform(globalState);
    Phi = (*TwoPi) * RandomNum;
    SinPhi = sin(Phi);
    CosPhi = cos(Phi);


    ARG1 = max(1.0 - S1*EI/COMEnergy[i],1E-20);

    D = 1.0 - CosTheta * sqrt(ARG1);
    U = (S1 - 1.0)*(S1-1.0)/ARG1;

    EBefore[i] = max(COMEnergy[i] * (1.0 - EI / (S1 * COMEnergy[i]) - 2.0 * D / S2), 1E-20);

    Q = min(sqrt((COMEnergy[i] / EBefore[i]) * ARG1) / S1,1.0);

    AngleFromZ[i] = asin(Q * sin(Theta));
    CosZAngle = cos(AngleFromZ[i]);

    if(CosTheta<0 && CosTheta*CosTheta>U){
      CosZAngle = -1 * CosZAngle;
    }
    SinZAngle = sin(AngleFromZ[i]);
    DZCOM = min(DZCOM,1.0);
    ARGZ = sqrt(DXCOM*DXCOM + DYCOM*DYCOM);
    if (ARGZ ==0){
      DirCosineZ1[i] = CosZAngle;
      DirCosineX1[i] = CosPhi * SinZAngle;
      DirCosineY1[i] = SinPhi * SinZAngle;
    }else{
      DirCosineZ1[i] = DZCOM * CosZAngle + ARGZ * SinZAngle * SinPhi;
      DirCosineY1[i] = DYCOM * CosZAngle + (SinZAngle / ARGZ) * (DXCOM * CosPhi - DYCOM * DZCOM * SinPhi);
      DirCosineX1[i] = DXCOM * CosZAngle - (SinZAngle / ARGZ) * (DYCOM * CosPhi + DXCOM * DZCOM * SinPhi);
    }

    CONST12 = (*Sqrt2M) * sqrt(EBefore[i]);
    VXLab = DirCosineX1[i] * CONST12 + GasVelX[i];
    VYLab = DirCosineY1[i] * CONST12 + GasVelY[i];
    VZLab = DirCosineZ1[i] * CONST12 + GasVelZ[i];

    EBefore[i] = (VXLab * VXLab + VYLab * VYLab + VZLab * VZLab) / (*TwoM);
    VelocityInCOM = ((*Sqrt2M) * sqrt(EBefore[i]));
    DirCosineX1[i] = VXLab / VelocityInCOM;
    DirCosineY1[i] = VYLab / VelocityInCOM;
    DirCosineZ1[i] = VZLab / VelocityInCOM;
}

// Copying constants into device
//Copying arrays to device


__global__ void MonteRun(double * EIN,double * ElectronEnergyStep,double *MaxCollisionFreqTotal,double * BP,double * F1,double * F2,double * Sqrt2M,
double * TwoM,double * TwoPi,double * ISize,double * NumPoints,double * MaxCollisionFreq,double * VTMB,double * X,double * Y,double * Z,double * TimeSum,
double * DirCosineZ1,double * DirCosineY1,double * DirCosineX1,double * EBefore,double * iEnergyBins,double * COMEnergy,double * VelocityX,double * VelocityY,
double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,double * T,double * AP,double * AngleFromZ,double * CF,double * ANGCT,double * SCA,
double * INDEX,double * IPN,double * RGAS,double * TotalCollisionFrequency,long long  * seeds, double * output
){
  int i = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  hiprandState state;
  hiprand_init(seeds[i], i, 0, &state);
  __syncthreads();

  int f = 0;

  for(int iColl=0;iColl<1000000;++iColl){
    GetCollisions(ElectronEnergyStep, MaxCollisionFreqTotal, BP,F1,
      F2,Sqrt2M,TwoM,TwoPi,MaxCollisionFreq, VTMB,TimeSum,
      DirCosineZ1, DirCosineX1, DirCosineY1, EBefore, iEnergyBins,
      COMEnergy, VelocityX, VelocityY,VelocityZ, GasVelX, GasVelY, GasVelZ,
      T, AP, TotalCollisionFrequency,i,&state);
      __syncthreads();

      ProcessCollisions(COMEnergy,VelocityX,VelocityY, VelocityZ, GasVelX,GasVelY, GasVelZ,
    AP, X, Y, Z, DirCosineX1,DirCosineY1,DirCosineZ1,iEnergyBins, CF, RGAS,EIN,
      INDEX,ANGCT, SCA, IPN, AngleFromZ,  TwoPi,  EBefore, Sqrt2M, TwoM,T,BP,F1,ISize,NumPoints,i,&state);
      if(((iColl)%(1000000/100))==0){

        output[0*100000+f*1000+i]=X[i];
        output[1*100000+f*1000+i]=Y[i];
        output[2*100000+f*1000+i]=Z[i];
        output[3*100000+f*1000+i]=TimeSum[i];
          f+=1;
      }
      __syncthreads();
  }
}

__global__ void Test(MonteGpuDevice * M){
  //printf("THIS IS IT %f\n",M->EnergyLevels[0]);
  M->Output[0] = 1000;

}
// function that will be called from the PyBoltz_Gpu class
void MonteGpu::MonteTGpu(){
  printf("HEREEEE %.10f\n", RGAS[10]);
  MonteGpuDevice * DeviceParameters = new MonteGpuDevice();


  MonteGpuDevice * DeviceParametersPointer;

  //DeviceParameters->EnergyLevels = LinearizeAndCopy2D(EnergyLevels,6,290);
  hipMemcpy(DeviceParameters->EnergyLevels,EnergyLevels,6*290*sizeof(double),hipMemcpyHostToDevice);

  printf("HERE\n");

  // Copying constants into device
  DeviceParameters->ElectronEnergyStep = SetupAndCopyDouble(&(ElectronEnergyStep),1);
  DeviceParameters->MaxCollisionFreqTotal = SetupAndCopyDouble(&(MaxCollisionFreqTotal),1);
  double bp = EField*EField*CONST1;
  DeviceParameters->BP = SetupAndCopyDouble(&(bp),1);
  double f1 = EField*CONST2;
  DeviceParameters->F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = EField*CONST3;
  DeviceParameters->F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = CONST3*0.01;
  DeviceParameters->Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  DeviceParameters->TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = pi*2;
  printf("HERE\n");

  DeviceParameters->TwoPi = SetupAndCopyDouble(&(twpi),1);
  DeviceParameters->ISIZE = SetupAndCopyDouble(ISIZE,6);
  printf("HERE\n");
  DeviceParameters->NumMomCrossSectionPoints = SetupAndCopyDouble(NumMomCrossSectionPoints,6);
  DeviceParameters->MaxCollisionFreq = SetupAndCopyDouble(MaxCollisionFreq,6);
  //Copying arrays to device
  DeviceParameters->VTMB = SetupAndCopyDouble((VTMB),6);
  DeviceParameters->X = SetupArrayOneVal(0,1000);
  DeviceParameters->Y = SetupArrayOneVal(0,1000);
  DeviceParameters->Z = SetupArrayOneVal(0,1000);
  DeviceParameters->TimeSum = SetupArrayOneVal(0,1000);
  DeviceParameters->DirCosineZ1 = SetupArrayOneVal(cos(AngleFromZ),1000);
  DeviceParameters->DirCosineX1 = SetupArrayOneVal(sin(AngleFromZ) * cos(AngleFromX),1000);
  DeviceParameters->DirCosineY1 = SetupArrayOneVal(sin(AngleFromZ) * sin(AngleFromX),1000);
  DeviceParameters->EBefore = SetupArrayOneVal(InitialElectronEnergy,1000);
  DeviceParameters->iEnergyBins = SetupArrayOneVal(0,1000);
  DeviceParameters->COMEnergy = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityX = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityY = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityZ = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelX = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelY = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelZ = SetupArrayOneVal(0,1000);
  printf("HERE\n");

  DeviceParameters->T = SetupArrayOneVal(0,1000);
  DeviceParameters->AP = SetupArrayOneVal(0,1000);
  hipMemcpy(DeviceParameters->TotalCollisionFrequency,TotalCollisionFrequency,6*4000*sizeof(double),hipMemcpyHostToDevice);

  DeviceParameters->AngleFromZ = SetupArrayOneVal(AngleFromZ,1000);
  hipMemcpy(DeviceParameters->CollisionFrequency,CollisionFrequency,6*4000*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->AngleCut,AngleCut,6*4000*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->ScatteringParameter,ScatteringParameter,6*4000*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->INDEX,INDEX,6*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->IPN,IPN,6*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->RGAS,RGAS,6*290*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(DeviceParameters->TotalCollisionFrequency,TotalCollisionFrequency,6*4000*sizeof(double),hipMemcpyHostToDevice);
  DeviceParameters->Output = SetupArrayOneVal(0,400000);
  printf("%.20f\n",sqrt2m*InitialElectronEnergy );

  srand(3);
  //RM48 stuff
  //struct RM48Gen* gen =(struct RM48Gen *)malloc(1000*sizeof(struct RM48Gen));
  long long * Seeds = (long long *)malloc(1000*sizeof(long long));
  for (int i=0;i<1000;i++){
    Seeds[i] = (rand()%100000000);
  }

  int f = 0;
  //printf("%d\n",gen[0].IJKLIN);

  long long * pointer;
  hipMalloc((void **)&pointer,1000*sizeof(long long));
  hipMemcpy(pointer,Seeds,1000*sizeof(long long),hipMemcpyHostToDevice);
  double * TT = (double *)malloc(1000*sizeof(double));

  hipMemcpy(DeviceParametersPointer,DeviceParameters,sizeof(MonteGpuDevice),hipMemcpyHostToDevice);

/*  MonteRun<<<25,40>>>(EIN, ElectronEnergyStep,MaxCollisionFreqTotal, BP, F1, F2, Sqrt2M,
   TwoM,TwoPi, ISize, NumPoints, MaxCollisionFreq, VTMB, X, Y, Z, TimeSum,
  DirCosineZ1, DirCosineY1, DirCosineX1, EBefore, iEnergyBins, COMEnergy, VelocityX, VelocityY,
   VelocityZ, GasVelX, GasVelY, GasVelZ, T, AP, AngleFromZ, CF, ANGCT, SCA,
   INDEX, IPN, RGAS, TotalCollisionFrequency, pointer, Output);*/
   Test<<<1,1>>>(DeviceParametersPointer);
   printf("HERE\n");
hipDeviceSynchronize();
   hipMemcpy(output,DeviceParameters->Output,400000*sizeof(double),hipMemcpyDeviceToHost);

   printf("THIS IS IT  %f\n",output[0]);

  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(DeviceParameters->Output);
  hipFree(pointer);
  hipFree(DeviceParameters->ElectronEnergyStep);
  hipFree(DeviceParameters->MaxCollisionFreqTotal);
  hipFree(DeviceParameters->BP);
  hipFree(DeviceParameters->F1);
  hipFree(DeviceParameters->F2);
  hipFree(DeviceParameters->Sqrt2M);
  hipFree(DeviceParameters->TwoM);
  hipFree(DeviceParameters->TwoPi);
  hipFree(DeviceParameters->ISIZE);
  hipFree(DeviceParameters->NumMomCrossSectionPoints);
  hipFree(DeviceParameters->MaxCollisionFreq);
  hipFree(DeviceParameters->VTMB);
  hipFree(DeviceParameters->X);
  hipFree(DeviceParameters->Y);
  hipFree(DeviceParameters->Z);
  hipFree(DeviceParameters->TimeSum);
  hipFree(DeviceParameters->DirCosineX1);
  hipFree(DeviceParameters->DirCosineY1);
  hipFree(DeviceParameters->DirCosineZ1);
  hipFree(DeviceParameters->EBefore);
  hipFree(DeviceParameters->iEnergyBins);
  hipFree(DeviceParameters->COMEnergy);
  hipFree(DeviceParameters->VelocityZ);
  hipFree(DeviceParameters->VelocityY);
  hipFree(DeviceParameters->VelocityX);
  hipFree(DeviceParameters->T);
  hipFree(DeviceParameters->GasVelX);
  hipFree(DeviceParameters->GasVelY);
  hipFree(DeviceParameters->GasVelZ);
  hipFree(DeviceParameters->AP);
  hipFree(DeviceParameters->AngleFromZ);
  hipFree(DeviceParameters->CollisionFrequency);
  hipFree(DeviceParameters->RGAS);
  hipFree(DeviceParameters->EnergyLevels);
  hipFree(DeviceParameters->AngleCut);
  hipFree(DeviceParameters->ScatteringParameter);
  hipFree(DeviceParameters->INDEX);
  hipFree(DeviceParameters->IPN);
  hipFree(DeviceParametersPointer);
}
