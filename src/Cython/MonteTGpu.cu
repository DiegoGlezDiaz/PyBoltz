#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include<hiprand/hiprand_kernel.h>
#include "MonteGpu.hh"
#define min(a,b) a<b?a:b
#define max(a,b) a>b?a:b



__device__ int MBSortT(long long GasIndex,double RandomNum,double iEnergyBin,double * CF,double ISIZE,double NumPoints){
  int ISTEP,INCR,I;
  ISTEP = ISIZE;
  INCR = 0;
  for(int K = 0;K<12;++K){
      I = INCR;
      if(ISTEP==2){
        if(I==0){
          return I;
        }
        return I - 1;
      }
      I = INCR + ISTEP/2;
      if (I<= NumPoints){
        if(CF[GasIndex*4000+(int)iEnergyBin*290+I]<RandomNum){
          INCR +=ISTEP;
        }
      }
      ISTEP = ISTEP/2;
  }
  if(I==0){
    return I;
  }
  return I - 1;
}


__device__ void GetCollisions(MonteGpuDevice * DP,int i,hiprandState* globalState){

  // function start
  int MaxBoltzNumsUsed = 0;
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = hiprand_uniform(globalState);
    R2 = hiprand_uniform(globalState);
    RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*DP->TwoPi)));
    RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*DP->TwoPi)));
  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = hiprand_uniform(globalState);
    DP->T[i] = -1 * log(RandomNum)/(*DP->MaxCollisionFreqTotal)+TDash;
    TDash = DP->T[i];
    DP->AP[i] = DP->DirCosineZ1[i]*(*DP->F2)*sqrt(DP->EBefore[i]);
    EAfter = DP->EBefore[i]+(DP->AP[i]+(*DP->BP)*DP->T[i])*DP->T[i];
    VelocityRatio = sqrt(DP->EBefore[i]/EAfter);
    DCosineZ2 = DP->DirCosineZ1[i] * VelocityRatio + DP->T[i] * (*DP->F2) / (2.0 * sqrt(EAfter));
    DCosineX2 = DP->DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DP->DirCosineY1[i] * VelocityRatio;
    MaxBoltzNumsUsed += 1;

    DP->GasIndex[i] = 0;

    RandomNum = hiprand_uniform(globalState);
    if(*DP->NumberOfGases==1){
      DP->GasIndex[i] = 0;
    }else{
      while(DP->MaxCollisionFreqTotalG[DP->GasIndex[i]]<RandomNum){
        DP->GasIndex[i]+=1;
      }
    }
    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){

        R1 = hiprand_uniform(globalState);
        R2 = hiprand_uniform(globalState);
        RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*DP->TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*DP->TwoPi)));
      }
      MaxBoltzNumsUsed = 1;
    }
    DP->GasVelX[i] = DP->VTMB[DP->GasIndex[i]] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    DP->GasVelY[i] = DP->VTMB[DP->GasIndex[i]] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    DP->GasVelZ[i] = DP->VTMB[DP->GasIndex[i]] * RNMX[MaxBoltzNumsUsed - 1];
    DP->VelocityX[i] = DCosineX2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->VelocityY[i] = DCosineY2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->VelocityZ[i] = DCosineZ2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->COMEnergy[i] = (pow((DP->VelocityX[i] - DP->GasVelX[i]), 2) + pow((DP->VelocityY[i] - DP->GasVelY[i]), 2) + pow(
        (DP->VelocityZ[i] - DP->GasVelZ[i]),
        2)) / (*DP->TwoM);
    DP->iEnergyBins[i] = DP->COMEnergy[i] / (*DP->ElectronEnergyStep);
    DP->iEnergyBins[i] = min(DP->iEnergyBins[i], 3999);
    RandomNum = hiprand_uniform(globalState);

    TEST = DP->TotalCollisionFrequency[DP->GasIndex[i]*4000+(int)DP->iEnergyBins[i]] / (DP->MaxCollisionFreq[DP->GasIndex[i]]);

    if (RandomNum < TEST){
      DP->TimeSum[i]+=DP->T[i];
      return;
    }else{
      // if here this is a null collision try again
      //Skipping null collision counters
    }

  }
}

__device__ void ProcessCollisions(MonteGpuDevice * DP,int i,hiprandState* globalState )
  {
    int I;
    double VelocityInCOM,DXCOM,DYCOM,DZCOM,T2,A,B,VelocityBefore,RandomNum;
    double S1,S2,EI,EXTRA,RandomNum2,CosTheta,EpsilonOkhr,Theta,Phi,SinPhi,CosPhi;
    double ARG1,D,U,Q,CosZAngle,SinZAngle,ARGZ,CONST12;
    double VXLab,VYLab,VZLab;
    VelocityInCOM  =  ((*DP->Sqrt2M) * sqrt(DP->COMEnergy[i]));

    DXCOM = (DP->VelocityX[i] - DP->GasVelX[i]) / VelocityInCOM;
    DYCOM = (DP->VelocityY[i] - DP->GasVelY[i]) / VelocityInCOM;
    DZCOM = (DP->VelocityZ[i] - DP->GasVelZ[i]) / VelocityInCOM;

    T2 = DP->T[i]*DP->T[i];
    A = DP->AP[i]*DP->T[i];
    B = (*DP->BP) * T2;

    VelocityBefore = (*DP->Sqrt2M) * sqrt(DP->EBefore[i]);

    A = DP->T[i] * VelocityBefore;
    DP->X[i] += DP->DirCosineX1[i] * A;
    DP->Y[i] += DP->DirCosineY1[i] * A;
    DP->Z[i] += DP->DirCosineZ1[i] * A + T2 * (*DP->F1);
    RandomNum = hiprand_uniform(globalState);

    I = MBSortT(DP->GasIndex[i],RandomNum,DP->iEnergyBins[i], DP->CollisionFrequency,(DP->ISIZE[0]),(DP->NumMomCrossSectionPoints[0]));
    while(DP->CollisionFrequency[DP->GasIndex[i]*4000+(int)DP->iEnergyBins[i]*290+I]<RandomNum) I+=1;


    S1 = DP->RGAS[DP->GasIndex[i]*290+I];
    EI = DP->EnergyLevels[DP->GasIndex[i]*290+I];

    if(DP->IPN[DP->GasIndex[i]*290+I]>0){
      RandomNum = hiprand_uniform(globalState);
      EXTRA = RandomNum * (DP->COMEnergy[i]-EI);
      EI = EXTRA + EI;
    }

    if(DP->COMEnergy[i]<EI){
      EI = DP->COMEnergy[i]-0.0001;
    }

    S2 = (S1*S1)/(S1 - 1.0);
    RandomNum = hiprand_uniform(globalState);

    if(DP->INDEX[DP->GasIndex[i]*290+I] == 1){
      RandomNum2 = hiprand_uniform(globalState);
      CosTheta = 1.0-RandomNum*DP->AngleCut[DP->GasIndex[i]*4000+(int)DP->iEnergyBins[i]*290 + I];
      if(RandomNum2>DP->ScatteringParameter[(DP->GasIndex[i]*4000+(int)DP->iEnergyBins[i]*290 + I)]){
        CosTheta = -1.0 * CosTheta;
      }
    }else if(DP->INDEX[DP->GasIndex[i]*290+I]==2){
      EpsilonOkhr = DP->ScatteringParameter[DP->GasIndex[i]*4000+(int)DP->iEnergyBins[i]*290 + I];
      CosTheta = 1.0 - (2.0 * RandomNum * (1.0 - EpsilonOkhr) / (1.0 + EpsilonOkhr * (1.0 - 2.0 * RandomNum)));
    }else{
      CosTheta = 1.0 - 2.0*RandomNum;
    }

    Theta = acos(CosTheta);
    RandomNum = hiprand_uniform(globalState);
    Phi = (*DP->TwoPi) * RandomNum;
    SinPhi = sin(Phi);
    CosPhi = cos(Phi);


    ARG1 = max(1.0 - S1*EI/DP->COMEnergy[i],1E-20);

    D = 1.0 - CosTheta * sqrt(ARG1);
    U = (S1 - 1.0)*(S1-1.0)/ARG1;

    DP->EBefore[i] = max(DP->COMEnergy[i] * (1.0 - EI / (S1 * DP->COMEnergy[i]) - 2.0 * D / S2), 1E-20);

    Q = min(sqrt((DP->COMEnergy[i] / DP->EBefore[i]) * ARG1) / S1,1.0);

    DP->AngleFromZ[i] = asin(Q * sin(Theta));
    CosZAngle = cos(DP->AngleFromZ[i]);

    if(CosTheta<0 && CosTheta*CosTheta>U){
      CosZAngle = -1 * CosZAngle;
    }
    SinZAngle = sin(DP->AngleFromZ[i]);
    DZCOM = min(DZCOM,1.0);
    ARGZ = sqrt(DXCOM*DXCOM + DYCOM*DYCOM);
    if (ARGZ ==0){
      DP->DirCosineZ1[i] = CosZAngle;
      DP->DirCosineX1[i] = CosPhi * SinZAngle;
      DP->DirCosineY1[i] = SinPhi * SinZAngle;
    }else{
      DP->DirCosineZ1[i] = DZCOM * CosZAngle + ARGZ * SinZAngle * SinPhi;
      DP->DirCosineY1[i] = DYCOM * CosZAngle + (SinZAngle / ARGZ) * (DXCOM * CosPhi - DYCOM * DZCOM * SinPhi);
      DP->DirCosineX1[i] = DXCOM * CosZAngle - (SinZAngle / ARGZ) * (DYCOM * CosPhi + DXCOM * DZCOM * SinPhi);
    }

    CONST12 = (*DP->Sqrt2M) * sqrt(DP->EBefore[i]);
    VXLab = DP->DirCosineX1[i] * CONST12 + DP->GasVelX[i];
    VYLab = DP->DirCosineY1[i] * CONST12 + DP->GasVelY[i];
    VZLab = DP->DirCosineZ1[i] * CONST12 + DP->GasVelZ[i];

    DP->EBefore[i] = (VXLab * VXLab + VYLab * VYLab + VZLab * VZLab) / (*DP->TwoM);
    VelocityInCOM = ((*DP->Sqrt2M) * sqrt(DP->EBefore[i]));
    DP->DirCosineX1[i] = VXLab / VelocityInCOM;
    DP->DirCosineY1[i] = VYLab / VelocityInCOM;
    DP->DirCosineZ1[i] = VZLab / VelocityInCOM;
}

// Copying constants into deviceDeviceParametersPointer
//Copying arrays to devicePointe


__global__ void MonteTRun(MonteGpuDevice * DP){
  int i = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  hiprandState state;
  hiprand_init(DP->SeedsGpu[i], i, 0, &state);
  __syncthreads();

  int f = 0;
  for(int iColl=0;iColl<(*DP->NumColls);++iColl){

      GetCollisions(DP,i,&state);
      __syncthreads();

      ProcessCollisions(DP,i,&state);
      if(((iColl)%(*DP->NumColls/100))==0){
        DP->XOutput[i*100+f]=DP->X[i];
        DP->YOutput[i*100+f]=DP->Y[i];
        DP->ZOutput[i*100+f]=DP->Z[i];
        DP->TimeSumOutput[i*100+f]=DP->TimeSum[i];
        f+=1;
      }
      __syncthreads();
  }
}

