#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include<hiprand/hiprand_kernel.h>
#include "MonteGpu.hh"
#define min(a,b) a<b?a:b
#define max(a,b) a>b?a:b



__device__ int MBSortT(double RandomNum,double iEnergyBin,double * CF,double ISIZE,double NumPoints){
  int ISTEP,INCR,I;
  ISTEP = ISIZE;
  INCR = 0;
  for(int K = 0;K<12;++K){
      I = INCR;
      if(ISTEP==2){
        if(I==0){
          return I;
        }
        return I - 1;
      }
      I = INCR + ISTEP/2;
      if (I<= NumPoints){
        if(CF[(int)iEnergyBin*290+I]<RandomNum){
          INCR +=ISTEP;
        }
      }
      ISTEP = ISTEP/2;
  }
  if(I==0){
    return I;
  }
  return I - 1;
}


__device__ extern void GetCollisions(MonteGpuDevice * DP,int i,hiprandState* globalState){

  // function start
  int MaxBoltzNumsUsed = 0;
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = hiprand_uniform(globalState);
    R2 = hiprand_uniform(globalState);
    RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*DP->TwoPi)));
    RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*DP->TwoPi)));
  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = hiprand_uniform(globalState);
    T[i] = -1 * log(RandomNum)/(*DP->MaxCollisionFreqTotal)+TDash;
    TDash = DP->T[i];
    AP[i] = DP->DirCosineZ1[i]*(*DP->F2)*sqrt(DP->EBefore[i]);
    EAfter = DP->EBefore[i]+(DP->AP[i]+(*DP->BP)*DP->T[i])*DP->T[i];
    VelocityRatio = sqrt(DP->EBefore[i]/EAfter);
    DCosineZ2 = DP->DirCosineZ1[i] * VelocityRatio + DP->T[i] * (*DP->F2) / (2.0 * sqrt(EAfter));
    DCosineX2 = DP->DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DP->DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;

    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){

        R1 = hiprand_uniform(globalState);
        R2 = hiprand_uniform(globalState);
        RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*DP->TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*DP->TwoPi)));
      }
      MaxBoltzNumsUsed = 1;
    }
    DP->GasVelX[i] = DP->VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    DP->GasVelY[i] = DP->VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    DP->GasVelZ[i] = DP->VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    DP->VelocityX[i] = DCosineX2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->VelocityY[i] = DCosineY2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->VelocityZ[i] = DCosineZ2 * (*DP->Sqrt2M) * sqrt(EAfter);
    DP->COMEnergy[i] = (pow((DP->VelocityX[i] - DP->GasVelX[i]), 2) + pow((DP->VelocityY[i] - DP->GasVelY[i]), 2) + pow(
        (DP->VelocityZ[i] - DP->GasVelZ[i]),
        2)) / (*DP->TwoM);
    DP->iEnergyBins[i] = DP->COMEnergy[i] / (*DP->ElectronEnergyStep);
    DP->iEnergyBins[i] = min(DP->iEnergyBins[i], 3999);
    RandomNum = hiprand_uniform(globalState);

    TEST = DP->TotalCollisionFrequency[(int)DP->iEnergyBins[i]] / (*DP->MaxCollisionFreq);
    if (RandomNum < TEST){
      DP->TimeSum[i]+=DP->T[i];
      return;
    }
  }
}

__device__ void ProcessCollisions(MonteGpuDevice * DP,int i,hiprandState* globalState )
  {
    int I;
    double VelocityInCOM,DXCOM,DYCOM,DZCOM,T2,A,B,VelocityBefore,RandomNum;
    double S1,S2,EI,EXTRA,RandomNum2,CosTheta,EpsilonOkhr,Theta,Phi,SinPhi,CosPhi;
    double ARG1,D,U,Q,CosZAngle,SinZAngle,ARGZ,CONST12;
    double VXLab,VYLab,VZLab;
    VelocityInCOM  =  ((*DP->Sqrt2M) * sqrt(DP->COMEnergy[i]));

    DXCOM = (DP->VelocityX[i] - DP->GasVelX[i]) / VelocityInCOM;
    DYCOM = (DP->VelocityY[i] - DP->GasVelY[i]) / VelocityInCOM;
    DZCOM = (DP->VelocityZ[i] - DP->GasVelZ[i]) / VelocityInCOM;

    T2 = DP->T[i]*DP->T[i];
    A = DP->AP[i]*DP->T[i];
    B = (*DP->BP) * T2;

    VelocityBefore = (*DP->Sqrt2M) * sqrt(DP->EBefore[i]);

    A = DP->T[i] * VelocityBefore;
    DP->X[i] += DP->DirCosineX1[i] * A;
    DP->Y[i] += DP->DirCosineY1[i] * A;
    DP->Z[i] += DP->DirCosineZ1[i] * A + T2 * (*DP->F1);
    RandomNum = hiprand_uniform(globalState);

    I = MBSortT(RandomNum,DP->iEnergyBin[i], DP->CollisionFrequency,(* DP->ISIZE),(*DP->NumMomCrossSectionPoints));
    while(DP->CollisionFrequency[(int)DP->iEnergyBin[i]*290+I]<RandomNum) I+=1;


    S1 = DP->RGAS[I];
    EI = DP->EnergyLevels[I];

    if(DP->IPN[I]>0){
      RandomNum = hiprand_uniform(globalState);
      EXTRA = RandomNum * (DP->COMEnergy[i]-EI);
      EI = EXTRA + EI;
    }

    if(DP->COMEnergy[i]<EI){
      EI = DP->COMEnergy[i]-0.0001;
    }

    S2 = (S1*S1)/(S1 - 1.0);
    RandomNum = hiprand_uniform(globalState);

    if(INDEX[I] == 1){
      RandomNum2 = hiprand_uniform(globalState);
      CosTheta = 1.0-RandomNum*DP->AngleCut[(int)iEnergyBin[i]*290 + I];
      if(RandomNum2>DP->ScatteringParameter[(int)DP->iEnergyBin[i]*290 + I]){
        CosTheta = -1.0 * CosTheta;
      }
    }else if(INDEX[I]==2){
      EpsilonOkhr = DP->ScatteringParameter[(int)DP->iEnergyBin[i]*290 + I];
      CosTheta = 1.0 - (2.0 * RandomNum * (1.0 - EpsilonOkhr) / (1.0 + EpsilonOkhr * (1.0 - 2.0 * RandomNum)));
    }else{
      CosTheta = 1.0 - 2.0*RandomNum;
    }

    Theta = acos(CosTheta);
    RandomNum = hiprand_uniform(globalState);
    Phi = (*DP->TwoPi) * RandomNum;
    SinPhi = sin(Phi);
    CosPhi = cos(Phi);


    ARG1 = max(1.0 - S1*EI/DP->COMEnergy[i],1E-20);

    D = 1.0 - CosTheta * sqrt(ARG1);
    U = (S1 - 1.0)*(S1-1.0)/ARG1;

    DP->EBefore[i] = max(DP->COMEnergy[i] * (1.0 - EI / (S1 * DP->COMEnergy[i]) - 2.0 * D / S2), 1E-20);

    Q = min(sqrt((DP->COMEnergy[i] / DP->EBefore[i]) * ARG1) / S1,1.0);

    DP->AngleFromZ[i] = asin(Q * sin(Theta));
    CosZAngle = cos(DP->AngleFromZ[i]);

    if(CosTheta<0 && CosTheta*CosTheta>U){
      CosZAngle = -1 * CosZAngle;
    }
    SinZAngle = sin(DP->AngleFromZ[i]);
    DZCOM = min(DZCOM,1.0);
    ARGZ = sqrt(DXCOM*DXCOM + DYCOM*DYCOM);
    if (ARGZ ==0){
      DP->DirCosineZ1[i] = CosZAngle;
      DP->DirCosineX1[i] = CosPhi * SinZAngle;
      DP->DirCosineY1[i] = SinPhi * SinZAngle;
    }else{
      DP->DirCosineZ1[i] = DZCOM * CosZAngle + ARGZ * SinZAngle * SinPhi;
      DP->DirCosineY1[i] = DYCOM * CosZAngle + (SinZAngle / ARGZ) * (DXCOM * CosPhi - DYCOM * DZCOM * SinPhi);
      DP->DirCosineX1[i] = DXCOM * CosZAngle - (SinZAngle / ARGZ) * (DYCOM * CosPhi + DXCOM * DZCOM * SinPhi);
    }

    CONST12 = (*DP->Sqrt2M) * sqrt(DP->EBefore[i]);
    VXLab = DP->DirCosineX1[i] * CONST12 + DP->GasVelX[i];
    VYLab = DP->DirCosineY1[i] * CONST12 + DP->GasVelY[i];
    VZLab = DP->DirCosineZ1[i] * CONST12 + DP->GasVelZ[i];

    DP->EBefore[i] = (VXLab * VXLab + VYLab * VYLab + VZLab * VZLab) / (*TwoM);
    VelocityInCOM = ((*DP->Sqrt2M) * sqrt(DP->EBefore[i]));
    DP->DirCosineX1[i] = VXLab / VelocityInCOM;
    DP->DirCosineY1[i] = VYLab / VelocityInCOM;
    DP->DirCosineZ1[i] = VZLab / VelocityInCOM;
}

// Copying constants into deviceDeviceParametersPointer
//Copying arrays to devicePointe


__global__ void MonteTRun(MonteGpuDevice * DP){
  int i = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  hiprandState state;
  hiprand_init(DP->SeedsGpu[i], i, 0, &state);
  __syncthreads();

  int f = 0;
  for(int iColl=0;iColl<DP->NumColls;++iColl){
    GetCollisions(DP,i,&state);
      __syncthreads();

      ProcessCollisions(DP,i,&state);
      if(((iColl)%(DP->NumColls/100))==0){
        DP->output[0*100000+f*1000+i]=DP->X[i];
        DP->output[1*100000+f*1000+i]=DP->Y[i];
        DP->output[2*100000+f*1000+i]=DP->Z[i];
        DP->output[3*100000+f*1000+i]=DP->TimeSum[i];
        f+=1;
      }
      __syncthreads();
  }
}

__global__ void Test(MonteGpuDevice * M,double * RGAS){

  M->Output[0] = 100;
  M->RGAS[10] = 100;
  __syncthreads();

}
// function that will be called from the PyBoltz_Gpu classoutput
void MonteGpu::MonteTGpu(){
  MonteGpuDevice * DeviceParametersPointer;

  hipMalloc((void **)&DeviceParametersPointer,sizeof(MonteGpuDevice));
  hipMemcpy(DeviceParametersPointer,DeviceParameters,sizeof(MonteGpuDevice),hipMemcpyHostToDevice);

  MonteTRun<<<threads,blocks>>>(DeviceParametersPointer);
  //Test<<<threads,blocks>>>(DeviceParametersPointer,DeviceParameters->RGAS);
  printf("HERE\n");
  hipDeviceSynchronize();
  hipMemcpy(DeviceParameters,DeviceParametersPointer,sizeof(MonteGpuDevice),hipMemcpyDeviceToHost);
  printf("%p\n", DeviceParameters->RGAS);
  hipMemcpy(output,DeviceParameters->Output,400000*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(RGAS,DeviceParameters->RGAS,6*290*sizeof(double),hipMemcpyDeviceToHost);
  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(DeviceParametersPointer);
}
