#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include<hiprand/hiprand_kernel.h>
#include "MonteGpu.hh"
#define min(a,b) a<b?a:b
#define max(a,b) a>b?a:b

// cudamalloc functions

double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

double * LinearizeAndCopy(double** arr,int h,int w){

  double * pointer;

  double * temp = (double *)malloc(h*w*sizeof(double));

  for(int i=0;i<h;++i){

    for(int j = 0;j<w;++j){
      temp[i*w+j] =arr[i][j];
    }
  }

  hipMalloc((void **)&pointer,h*w*sizeof(double));
  hipMemcpy(pointer,temp,h*w*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

__device__ int MBSortT(double RandomNum,double iEnergyBin,double * CF,double ISIZE,double NumPoints){
  int ISTEP,INCR,I;
  ISTEP = ISIZE;
  INCR = 0;
  for(int K = 0;K<12;++K){
      I = INCR;
      if(ISTEP==2){
        if(I==0){
          return I;
        }
        return I - 1;
      }
      I = INCR + ISTEP/2;
      if (I<= NumPoints){
        if(CF[(int)iEnergyBin*290+I]<RandomNum){
          INCR +=ISTEP;
        }
      }
      ISTEP = ISTEP/2;
  }
  if(I==0){
    return I;
  }
  return I - 1;
}


__device__ extern void GetCollisions(double *ElectronEnergyStep, double* MaxCollisionFreqTotal,double* BP,double*  F1,
  double*  F2,double* Sqrt2M,double* TwoM,double* TwoPi,double* MaxCollisionFreq,double * VTMB,double * TimeSum,
  double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
  double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * T,double * AP,double * TotalCollisionFrequency,int i,hiprandState* globalState){

  // function start
  int MaxBoltzNumsUsed = 0;
  hiprand_uniform(globalState);
  //R = hiprand_uniform( &state );
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = hiprand_uniform(globalState);
    R2 = hiprand_uniform(globalState);
    RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*TwoPi)));
    RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*TwoPi)));
  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = hiprand_uniform(globalState);
    T[i] = -1 * log(RandomNum)/(*MaxCollisionFreqTotal)+TDash;
    TDash = T[i];
    AP[i] = DirCosineZ1[i]*(*F2)*sqrt(EBefore[i]);
    EAfter = EBefore[i]+(AP[i]+(*BP)*T[i])*T[i];
    VelocityRatio = sqrt(EBefore[i]/EAfter);
    DCosineZ2 = DirCosineZ1[i] * VelocityRatio + T[i] * (*F2) / (2.0 * sqrt(EAfter));
    DCosineX2 = DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;

    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){
        R1 = hiprand_uniform(globalState);
        R2 = hiprand_uniform(globalState);
        RNMX[j] = sqrt(-1*log(R1))*cos(R2*((*TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*((*TwoPi)));
      }
      MaxBoltzNumsUsed = 1;
    }
    GasVelX[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelY[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelZ[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    VelocityX[i] = DCosineX2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityY[i] = DCosineY2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityZ[i] = DCosineZ2 * (*Sqrt2M) * sqrt(EAfter);
    COMEnergy[i] = (pow((VelocityX[i] - GasVelX[i]), 2) + pow((VelocityY[i] - GasVelY[i]), 2) + pow(
        (VelocityZ[i] - GasVelZ[i]),
        2)) / (*TwoM);
    iEnergyBins[i] = COMEnergy[i] / (*ElectronEnergyStep);
    iEnergyBins[i] = min(iEnergyBins[i], 3999);
    RandomNum = hiprand_uniform(globalState);

    TEST = TotalCollisionFrequency[(int)iEnergyBins[i]] / (*MaxCollisionFreq);
    if (RandomNum < TEST){
      TimeSum[i]+=T[i];
      return;
    }
  }
}

__device__ void ProcessCollisions(double *COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * AP, double * X,double * Y,double * Z, double *DirCosineX1,double *DirCosineY1,double *DirCosineZ1,double * iEnergyBin,double * CF,double * RGAS,double * EnergyLevels,
  double * INDEX, double * ANGCT, double * SCA, double * IPN, double * AngleFromZ, double * TwoPi, double * EBefore, double * Sqrt2M,
  double * TwoM,double *T,double * BP,double * F1,double * ISIZE,double * NumPoints,int i,hiprandState* globalState )
  {
    int I;
    double VelocityInCOM,DXCOM,DYCOM,DZCOM,T2,A,B,VelocityBefore,RandomNum;
    double S1,S2,EI,EXTRA,RandomNum2,CosTheta,EpsilonOkhr,Theta,Phi,SinPhi,CosPhi;
    double ARG1,D,U,Q,CosZAngle,SinZAngle,ARGZ,CONST12;
    double VXLab,VYLab,VZLab;
    VelocityInCOM  =  ((*Sqrt2M) * sqrt(COMEnergy[i]));

    DXCOM = (VelocityX[i] - GasVelX[i]) / VelocityInCOM;
    DYCOM = (VelocityY[i] - GasVelY[i]) / VelocityInCOM;
    DZCOM = (VelocityZ[i] - GasVelZ[i]) / VelocityInCOM;

    T2 = T[i]*T[i];
    A = AP[i]*T[i];
    B = (*BP) * T2;

    VelocityBefore = (*Sqrt2M) * sqrt(EBefore[i]);

    A = T[i] * VelocityBefore;
    X[i] += DirCosineX1[i] * A;
    Y[i] += DirCosineY1[i] * A;
    Z[i] += DirCosineZ1[i] * A + T2 * (*F1);
    RandomNum = hiprand_uniform(globalState);

    I = MBSortT(RandomNum,iEnergyBin[i], CF,(*ISIZE),(*NumPoints));
    while(CF[(int)iEnergyBin[i]*290+I]<RandomNum) I+=1;


    S1 = RGAS[I];
    EI = EnergyLevels[I];

    if(IPN[I]>0){
      RandomNum = hiprand_uniform(globalState);
      EXTRA = RandomNum * (COMEnergy[i]-EI);
      EI = EXTRA + EI;
    }

    if(COMEnergy[i]<EI){
      EI = COMEnergy[i]-0.0001;
    }

    S2 = (S1*S1)/(S1 - 1.0);
    RandomNum = hiprand_uniform(globalState);

    if(INDEX[I] == 1){
      RandomNum2 = hiprand_uniform(globalState);
      CosTheta = 1.0-RandomNum*ANGCT[(int)iEnergyBin[i]*290 + I];
      if(RandomNum2>SCA[(int)iEnergyBin[i]*290 + I]){
        CosTheta = -1.0 * CosTheta;
      }
    }else if(INDEX[I]==2){
      EpsilonOkhr = SCA[(int)iEnergyBin[i]*290 + I];
      CosTheta = 1.0 - (2.0 * RandomNum * (1.0 - EpsilonOkhr) / (1.0 + EpsilonOkhr * (1.0 - 2.0 * RandomNum)));
    }else{
      CosTheta = 1.0 - 2.0*RandomNum;
    }

    Theta = acos(CosTheta);
    RandomNum = hiprand_uniform(globalState);
    Phi = (*TwoPi) * RandomNum;
    SinPhi = sin(Phi);
    CosPhi = cos(Phi);


    ARG1 = max(1.0 - S1*EI/COMEnergy[i],1E-20);

    D = 1.0 - CosTheta * sqrt(ARG1);
    U = (S1 - 1.0)*(S1-1.0)/ARG1;

    EBefore[i] = max(COMEnergy[i] * (1.0 - EI / (S1 * COMEnergy[i]) - 2.0 * D / S2), 1E-20);

    Q = min(sqrt((COMEnergy[i] / EBefore[i]) * ARG1) / S1,1.0);

    AngleFromZ[i] = asin(Q * sin(Theta));
    CosZAngle = cos(AngleFromZ[i]);

    if(CosTheta<0 && CosTheta*CosTheta>U){
      CosZAngle = -1 * CosZAngle;
    }
    SinZAngle = sin(AngleFromZ[i]);
    DZCOM = min(DZCOM,1.0);
    ARGZ = sqrt(DXCOM*DXCOM + DYCOM*DYCOM);
    if (ARGZ ==0){
      DirCosineZ1[i] = CosZAngle;
      DirCosineX1[i] = CosPhi * SinZAngle;
      DirCosineY1[i] = SinPhi * SinZAngle;
    }else{
      DirCosineZ1[i] = DZCOM * CosZAngle + ARGZ * SinZAngle * SinPhi;
      DirCosineY1[i] = DYCOM * CosZAngle + (SinZAngle / ARGZ) * (DXCOM * CosPhi - DYCOM * DZCOM * SinPhi);
      DirCosineX1[i] = DXCOM * CosZAngle - (SinZAngle / ARGZ) * (DYCOM * CosPhi + DXCOM * DZCOM * SinPhi);
    }

    CONST12 = (*Sqrt2M) * sqrt(EBefore[i]);
    VXLab = DirCosineX1[i] * CONST12 + GasVelX[i];
    VYLab = DirCosineY1[i] * CONST12 + GasVelY[i];
    VZLab = DirCosineZ1[i] * CONST12 + GasVelZ[i];

    EBefore[i] = (VXLab * VXLab + VYLab * VYLab + VZLab * VZLab) / (*TwoM);
    VelocityInCOM = ((*Sqrt2M) * sqrt(EBefore[i]));
    DirCosineX1[i] = VXLab / VelocityInCOM;
    DirCosineY1[i] = VYLab / VelocityInCOM;
    DirCosineZ1[i] = VZLab / VelocityInCOM;
}

// Copying constants into device
//Copying arrays to device


__global__ void MonteRun(double * EIN,double * ElectronEnergyStep,double *MaxCollisionFreqTotal,double * BP,double * F1,double * F2,double * Sqrt2M,
double * TwoM,double * TwoPi,double * ISize,double * NumPoints,double * MaxCollisionFreq,double * VTMB,double * X,double * Y,double * Z,double * TimeSum,
double * DirCosineZ1,double * DirCosineY1,double * DirCosineX1,double * EBefore,double * iEnergyBins,double * COMEnergy,double * VelocityX,double * VelocityY,
double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,double * T,double * AP,double * AngleFromZ,double * CF,double * ANGCT,double * SCA,
double * INDEX,double * IPN,double * RGAS,double * TotalCollisionFrequency,long long  * seeds, double * output
){
  int i = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  hiprandState state;
  hiprand_init(seeds[i], i, 0, &state);
  __syncthreads();

  int f = 0;

  for(int iColl=0;iColl<1000000;++iColl){
    GetCollisions(ElectronEnergyStep, MaxCollisionFreqTotal, BP,F1,
      F2,Sqrt2M,TwoM,TwoPi,MaxCollisionFreq, VTMB,TimeSum,
      DirCosineZ1, DirCosineX1, DirCosineY1, EBefore, iEnergyBins,
      COMEnergy, VelocityX, VelocityY,VelocityZ, GasVelX, GasVelY, GasVelZ,
      T, AP, TotalCollisionFrequency,i,&state);
      __syncthreads();

      ProcessCollisions(COMEnergy,VelocityX,VelocityY, VelocityZ, GasVelX,GasVelY, GasVelZ,
    AP, X, Y, Z, DirCosineX1,DirCosineY1,DirCosineZ1,iEnergyBins, CF, RGAS,EIN,
      INDEX,ANGCT, SCA, IPN, AngleFromZ,  TwoPi,  EBefore, Sqrt2M, TwoM,T,BP,F1,ISize,NumPoints,i,&state);
      if(((iColl)%(1000000/100))==0){

        output[0*100000+f*1000+i]=X[i];
        output[1*100000+f*1000+i]=Y[i];
        output[2*100000+f*1000+i]=Z[i];
        output[3*100000+f*1000+i]=TimeSum[i];
          f+=1;
      }
      __syncthreads();
  }
}


// function that will be called from the PyBoltz_Gpu class
void MonteGpu::MonteTGpu(){
  printf("HEREEEE %f\n", PElectronEnergyStep);

  double * EIN = LinearizeAndCopy(PEnergyLevels,6,290);
  // Copying constants into device
  double * ElectronEnergyStep = SetupAndCopyDouble(&(PElectronEnergyStep),1);
  double * MaxCollisionFreqTotal = SetupAndCopyDouble(&(PMaxCollisionFreqTotal),1);
  double bp = PEField*PEField*PCONST1;
  double * BP = SetupAndCopyDouble(&(bp),1);
  double f1 = PEField*PCONST2;
  double * F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = PEField*PCONST3;
  double * F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = PCONST3*0.01;
  double * Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  double * TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = Ppi*2;
  double * TwoPi = SetupAndCopyDouble(&(twpi),1);
  double * ISize = SetupAndCopyDouble(&(PISIZE[0]),1);
  double * NumPoints = SetupAndCopyDouble(&(PNumMomCrossSectionPoints),1);
  double * MaxCollisionFreq = SetupAndCopyDouble(&(PMaxCollisionFreq),1);

  //Copying arrays to device
  double * VTMB = SetupAndCopyDouble((PVTMB),6);
  double * X = SetupArrayOneVal(0,1000);
  double * Y = SetupArrayOneVal(0,1000);
  double * Z = SetupArrayOneVal(0,1000);
  double * TimeSum = SetupArrayOneVal(0,1000);
  double * DirCosineZ1 = SetupArrayOneVal(cos(PAngleFromZ),1000);
  double * DirCosineX1 = SetupArrayOneVal(sin(PAngleFromZ) * cos(PAngleFromX),1000);
  double * DirCosineY1 = SetupArrayOneVal(sin(PAngleFromZ) * sin(PAngleFromX),1000);
  double * EBefore = SetupArrayOneVal(PInitialElectronEnergy,1000);
  double * iEnergyBins = SetupArrayOneVal(0,1000);
  double * COMEnergy = SetupArrayOneVal(0,1000);
  double * VelocityX = SetupArrayOneVal(0,1000);
  double * VelocityY = SetupArrayOneVal(0,1000);
  double * VelocityZ = SetupArrayOneVal(0,1000);
  double * GasVelX = SetupArrayOneVal(0,1000);
  double * GasVelY = SetupArrayOneVal(0,1000);
  double * GasVelZ = SetupArrayOneVal(0,1000);
  double * T = SetupArrayOneVal(0,1000);
  double * AP = SetupArrayOneVal(0,1000);
  double * AngleFromZ = SetupArrayOneVal(PAngleFromZ,1000);
  double * CF = LinearizeAndCopy((double **)PCollisionFrequency,4000,290);
  double * ANGCT = LinearizeAndCopy((double **)PAngleCut,4000,290);
  double * SCA = LinearizeAndCopy((double **)PScatteringParameter,4000,290);
  double * INDEX = SetupAndCopyDouble((PINDEX),290);
  double * IPN = SetupAndCopyDouble((PIPN),290);
  double * RGAS = LinearizeAndCopy((double **)PRGAS,6,290);
  double * Output = SetupArrayOneVal(0,400000);
  double * TotalCollisionFrequency = SetupAndCopyDouble(PTotalCollisionFrequency,4000);
  printf("%.20f\n",sqrt2m*PInitialElectronEnergy );

  srand(3);
  //RM48 stuff
  //struct RM48Gen* gen =(struct RM48Gen *)malloc(1000*sizeof(struct RM48Gen));
  long long * Seeds = (long long *)malloc(1000*sizeof(long long));
  for (int i=0;i<1000;i++){
    Seeds[i] = (rand()%100000000);
  }

  int f = 0;
  //printf("%d\n",gen[0].IJKLIN);

  long long * pointer;
  hipMalloc((void **)&pointer,1000*sizeof(long long));
  hipMemcpy(pointer,Seeds,1000*sizeof(long long),hipMemcpyHostToDevice);
  double * TT = (double *)malloc(1000*sizeof(double));

  MonteRun<<<25,40>>>(EIN, ElectronEnergyStep,MaxCollisionFreqTotal, BP, F1, F2, Sqrt2M,
   TwoM,TwoPi, ISize, NumPoints, MaxCollisionFreq, VTMB, X, Y, Z, TimeSum,
  DirCosineZ1, DirCosineY1, DirCosineX1, EBefore, iEnergyBins, COMEnergy, VelocityX, VelocityY,
   VelocityZ, GasVelX, GasVelY, GasVelZ, T, AP, AngleFromZ, CF, ANGCT, SCA,
   INDEX, IPN, RGAS, TotalCollisionFrequency, pointer, Output);
   hipMemcpy(output,Output,400000*sizeof(double),hipMemcpyDeviceToHost);
  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(Output);
  hipFree(pointer);
  hipFree(ElectronEnergyStep);
  hipFree(MaxCollisionFreqTotal);
  hipFree(BP);
  hipFree(F1);
  hipFree(F2);
  hipFree(Sqrt2M);
  hipFree(TwoM);
  hipFree(TwoPi);
  hipFree(ISize);
  hipFree(NumPoints);
  hipFree(MaxCollisionFreq);
  hipFree(VTMB);
  hipFree(X);
  hipFree(Y);
  hipFree(Z);
  hipFree(TimeSum);
  hipFree(DirCosineX1);
  hipFree(DirCosineY1);
  hipFree(DirCosineZ1);
  hipFree(EBefore);
  hipFree(iEnergyBins);
  hipFree(COMEnergy);
  hipFree(VelocityZ);
  hipFree(VelocityY);
  hipFree(VelocityX);
  hipFree(T);
  hipFree(GasVelX);
  hipFree(GasVelY);
  hipFree(GasVelZ);
  hipFree(AP);
  hipFree(AngleFromZ);
  hipFree(CF);
  hipFree(RGAS);
  hipFree(EIN);
  hipFree(ANGCT);
  hipFree(SCA);
  hipFree(INDEX);
  hipFree(IPN);
}
