#include "hip/hip_runtime.h"
#include "MonteGpu.hh"
#include <stdio.h>


// cudamalloc functions

double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

long long *SetupAndCopyllong(long long * data,int s){
  long long * pointer;
  hipMalloc((void **)&pointer,s*sizeof(long long));
  hipMemcpy(pointer,data,s*sizeof(long long),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

void MonteGpu::Setup(){
  DeviceParameters = new MonteGpuDevice();
  DeviceParameters->EnergyLevels = SetupAndCopyDouble(EnergyLevels,6*290);
  // Copying constants into device
  DeviceParameters->ElectronEnergyStep = SetupAndCopyDouble(&(ElectronEnergyStep),1);
  DeviceParameters->MaxCollisionFreqTotal = SetupAndCopyDouble(&(MaxCollisionFreqTotal),1);
  double bp = EField*EField*CONST1;
  DeviceParameters->BP = SetupAndCopyDouble(&(bp),1);
  double f1 = EField*CONST2;
  DeviceParameters->F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = EField*CONST3;
  DeviceParameters->F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = CONST3*0.01;
  DeviceParameters->Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  DeviceParameters->TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = pi*2;
  DeviceParameters->TwoPi = SetupAndCopyDouble(&(twpi),1);
  DeviceParameters->SeedsGpu = SetupAndCopyllong(SeedsGpu,numElectrons);
  DeviceParameters->NumColls = SetupAndCopyllong(&NumColls,1);
  DeviceParameters->TwoPi = SetupAndCopyDouble(&(twpi),1);
  DeviceParameters->ISIZE = SetupAndCopyDouble(ISIZE,6);
  DeviceParameters->NumMomCrossSectionPoints = SetupAndCopyDouble(NumMomCrossSectionPoints,6);
  DeviceParameters->MaxCollisionFreq = SetupAndCopyDouble(MaxCollisionFreq,6);
  //Copying arrays to device
  DeviceParameters->VTMB = SetupAndCopyDouble((VTMB),6);
  DeviceParameters->X = SetupArrayOneVal(0,1000);
  DeviceParameters->Y = SetupArrayOneVal(0,1000);
  DeviceParameters->Z = SetupArrayOneVal(0,1000);
  DeviceParameters->TimeSum = SetupArrayOneVal(0,1000);
  DeviceParameters->DirCosineZ1 = SetupArrayOneVal(cos(AngleFromZ),1000);
  DeviceParameters->DirCosineX1 = SetupArrayOneVal(sin(AngleFromZ) * cos(AngleFromX),1000);
  DeviceParameters->DirCosineY1 = SetupArrayOneVal(sin(AngleFromZ) * sin(AngleFromX),1000);
  DeviceParameters->EBefore = SetupArrayOneVal(InitialElectronEnergy,1000);
  DeviceParameters->iEnergyBins = SetupArrayOneVal(0,1000);
  DeviceParameters->COMEnergy = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityX = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityY = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityZ = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelX = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelY = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelZ = SetupArrayOneVal(0,1000);
  DeviceParameters->T = SetupArrayOneVal(0,1000);
  DeviceParameters->AP = SetupArrayOneVal(0,1000);
  DeviceParameters->AngleFromZ = SetupArrayOneVal(AngleFromZ,1000);
  DeviceParameters->CollisionFrequency = SetupAndCopyDouble(CollisionFrequency,6*4000*290);
  DeviceParameters->AngleCut = SetupAndCopyDouble(AngleCut,6*4000*290);
  DeviceParameters->ScatteringParameter = SetupAndCopyDouble(ScatteringParameter,6*4000*290);
  DeviceParameters->INDEX = SetupAndCopyDouble(INDEX,6*290);
  DeviceParameters->IPN = SetupAndCopyDouble(IPN,6*290);
  DeviceParameters->RGAS = SetupAndCopyDouble(RGAS,6*290);
  DeviceParameters->TotalCollisionFrequency = SetupAndCopyDouble(TotalCollisionFrequency,6*4000);
  DeviceParameters->Output = SetupArrayOneVal(0,400000);
}

MonteGpu::~MonteGpu(){
  hipFree(DeviceParameters->ElectronEnergyStep);
  hipFree(DeviceParameters->MaxCollisionFreqTotal);
  hipFree(DeviceParameters->BP);
  hipFree(DeviceParameters->F1);
  hipFree(DeviceParameters->F2);
  hipFree(DeviceParameters->Sqrt2M);
  hipFree(DeviceParameters->TwoM);
  hipFree(DeviceParameters->TwoPi);
  hipFree(DeviceParameters->ISIZE);
  hipFree(DeviceParameters->NumMomCrossSectionPoints);
  hipFree(DeviceParameters->MaxCollisionFreq);
  hipFree(DeviceParameters->VTMB);
  hipFree(DeviceParameters->SeedsGpu);
  hipFree(DeviceParameters->NumColls);
  hipFree(DeviceParameters->X);
  hipFree(DeviceParameters->Y);
  hipFree(DeviceParameters->Z);
  hipFree(DeviceParameters->TimeSum);
  hipFree(DeviceParameters->DirCosineX1);
  hipFree(DeviceParameters->DirCosineY1);
  hipFree(DeviceParameters->DirCosineZ1);
  hipFree(DeviceParameters->EBefore);
  hipFree(DeviceParameters->iEnergyBins);
  hipFree(DeviceParameters->COMEnergy);
  hipFree(DeviceParameters->VelocityZ);
  hipFree(DeviceParameters->VelocityY);
  hipFree(DeviceParameters->VelocityX);
  hipFree(DeviceParameters->T);
  hipFree(DeviceParameters->GasVelX);
  hipFree(DeviceParameters->GasVelY);
  hipFree(DeviceParameters->GasVelZ);
  hipFree(DeviceParameters->AP);
  hipFree(DeviceParameters->AngleFromZ);
  hipFree(DeviceParameters->CollisionFrequency);
  hipFree(DeviceParameters->RGAS);
  hipFree(DeviceParameters->EnergyLevels);
  hipFree(DeviceParameters->AngleCut);
  hipFree(DeviceParameters->ScatteringParameter);
  hipFree(DeviceParameters->INDEX);
  hipFree(DeviceParameters->IPN);
  hipFree(DeviceParameters->Output);
}
