#include "hip/hip_runtime.h"
#include "MonteGpu.hh"
#include <stdio.h>

extern __global__ void MonteTRun(MonteGpuDevice * DP);

// cudamalloc functions

double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

long long *SetupAndCopyllong(long long * data,int s){
  long long * pointer;
  hipMalloc((void **)&pointer,s*sizeof(long long));
  hipMemcpy(pointer,data,s*sizeof(long long),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

long long * SetupArrayOneValLL(long long val,int s){
  long long * pointer;
  hipMalloc((void **)&pointer,s*sizeof(long long));
  long long * temp = (long long *)malloc(s*sizeof(long long));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(long long),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}

void MonteGpu::Setup(){
  DeviceParameters = new MonteGpuDevice();
  DeviceParameters->EnergyLevels = SetupAndCopyDouble(EnergyLevels,6*290);
  // Copying constants into device
  DeviceParameters->ElectronEnergyStep = SetupAndCopyDouble(&(ElectronEnergyStep),1);
  DeviceParameters->MaxCollisionFreqTotal = SetupAndCopyDouble(&(MaxCollisionFreqTotal),1);
  double bp = EField*EField*CONST1;
  DeviceParameters->BP = SetupAndCopyDouble(&(bp),1);
  double f1 = EField*CONST2;
  DeviceParameters->F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = EField*CONST3;
  DeviceParameters->F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = CONST3*0.01;
  DeviceParameters->Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  DeviceParameters->TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = pi*2;
  DeviceParameters->TwoPi = SetupAndCopyDouble(&(twpi),1);
  DeviceParameters->SeedsGpu = SetupAndCopyllong(SeedsGpu,numElectrons);
  DeviceParameters->NumColls = SetupAndCopyllong(&NumColls,1);
  DeviceParameters->TwoPi = SetupAndCopyDouble(&(twpi),1);
  DeviceParameters->ISIZE = SetupAndCopyDouble(ISIZE,6);
  DeviceParameters->NumMomCrossSectionPoints = SetupAndCopyDouble(NumMomCrossSectionPoints,6);
  DeviceParameters->MaxCollisionFreq = SetupAndCopyDouble(MaxCollisionFreq,6);
  //Copying arrays to device
  DeviceParameters->VTMB = SetupAndCopyDouble((VTMB),6);
  DeviceParameters->X = SetupArrayOneVal(0,1000);
  DeviceParameters->Y = SetupArrayOneVal(0,1000);
  DeviceParameters->Z = SetupArrayOneVal(0,1000);
  DeviceParameters->TimeSum = SetupArrayOneVal(0,1000);
  DeviceParameters->DirCosineZ1 = SetupArrayOneVal(cos(AngleFromZ),1000);
  DeviceParameters->DirCosineX1 = SetupArrayOneVal(sin(AngleFromZ) * cos(AngleFromX),1000);
  DeviceParameters->DirCosineY1 = SetupArrayOneVal(sin(AngleFromZ) * sin(AngleFromX),1000);
  DeviceParameters->EBefore = SetupArrayOneVal(InitialElectronEnergy,1000);
  DeviceParameters->iEnergyBins = SetupArrayOneVal(0,1000);
  DeviceParameters->COMEnergy = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityX = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityY = SetupArrayOneVal(0,1000);
  DeviceParameters->VelocityZ = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelX = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelY = SetupArrayOneVal(0,1000);
  DeviceParameters->GasVelZ = SetupArrayOneVal(0,1000);
  DeviceParameters->T = SetupArrayOneVal(0,1000);
  DeviceParameters->AP = SetupArrayOneVal(0,1000);
  DeviceParameters->AngleFromZ = SetupArrayOneVal(AngleFromZ,1000);
  DeviceParameters->CollisionFrequency = SetupAndCopyDouble(CollisionFrequency,6*4000*290);
  DeviceParameters->AngleCut = SetupAndCopyDouble(AngleCut,6*4000*290);
  DeviceParameters->ScatteringParameter = SetupAndCopyDouble(ScatteringParameter,6*4000*290);
  DeviceParameters->INDEX = SetupAndCopyDouble(INDEX,6*290);
  DeviceParameters->IPN = SetupAndCopyDouble(IPN,6*290);
  DeviceParameters->RGAS = SetupAndCopyDouble(RGAS,6*290);
  DeviceParameters->TotalCollisionFrequency = SetupAndCopyDouble(TotalCollisionFrequency,6*4000);
  DeviceParameters->XOutput = SetupArrayOneVal(0,100000);
  DeviceParameters->YOutput = SetupArrayOneVal(0,100000);
  DeviceParameters->ZOutput = SetupArrayOneVal(0,100000);
  DeviceParameters->TimeSumOutput = SetupArrayOneVal(0,100000);
  DeviceParameters->GasIndex = SetupArrayOneValLL(0,1000);
  DeviceParameters->MaxCollisionFreqTotalG = SetupAndCopyDouble(MaxCollisionFreqTotalG,6);
  DeviceParameters->NumberOfGases =  SetupAndCopyDouble(&(NumberOfGases),1);
}

MonteGpu::~MonteGpu(){
  hipFree(DeviceParameters->ElectronEnergyStep);
  hipFree(DeviceParameters->MaxCollisionFreqTotal);
  hipFree(DeviceParameters->BP);
  hipFree(DeviceParameters->F1);
  hipFree(DeviceParameters->F2);
  hipFree(DeviceParameters->Sqrt2M);
  hipFree(DeviceParameters->TwoM);
  hipFree(DeviceParameters->TwoPi);
  hipFree(DeviceParameters->ISIZE);
  hipFree(DeviceParameters->NumMomCrossSectionPoints);
  hipFree(DeviceParameters->MaxCollisionFreq);
  hipFree(DeviceParameters->VTMB);
  hipFree(DeviceParameters->SeedsGpu);
  hipFree(DeviceParameters->NumColls);
  hipFree(DeviceParameters->X);
  hipFree(DeviceParameters->Y);
  hipFree(DeviceParameters->Z);
  hipFree(DeviceParameters->TimeSum);
  hipFree(DeviceParameters->DirCosineX1);
  hipFree(DeviceParameters->DirCosineY1);
  hipFree(DeviceParameters->DirCosineZ1);
  hipFree(DeviceParameters->EBefore);
  hipFree(DeviceParameters->iEnergyBins);
  hipFree(DeviceParameters->COMEnergy);
  hipFree(DeviceParameters->VelocityZ);
  hipFree(DeviceParameters->VelocityY);
  hipFree(DeviceParameters->VelocityX);
  hipFree(DeviceParameters->T);
  hipFree(DeviceParameters->GasVelX);
  hipFree(DeviceParameters->GasVelY);
  hipFree(DeviceParameters->GasVelZ);
  hipFree(DeviceParameters->AP);
  hipFree(DeviceParameters->AngleFromZ);
  hipFree(DeviceParameters->CollisionFrequency);
  hipFree(DeviceParameters->RGAS);
  hipFree(DeviceParameters->EnergyLevels);
  hipFree(DeviceParameters->AngleCut);
  hipFree(DeviceParameters->ScatteringParameter);
  hipFree(DeviceParameters->INDEX);
  hipFree(DeviceParameters->IPN);
  hipFree(DeviceParameters->XOutput);
  hipFree(DeviceParameters->YOutput);
  hipFree(DeviceParameters->ZOutput);
  hipFree(DeviceParameters->TimeSumOutput);
  hipFree(DeviceParameters->GasIndex);
  hipFree(DeviceParameters->NumberOfGases);
  hipFree(DeviceParameters->MaxCollisionFreqTotalG);
}

// function that will be called from the PyBoltz_Gpu classoutput
void MonteGpu::MonteRunGpu(){
  MonteGpuDevice * DeviceParametersPointer;

  hipMalloc((void **)&DeviceParametersPointer,sizeof(MonteGpuDevice));
  hipMemcpy(DeviceParametersPointer,DeviceParameters,sizeof(MonteGpuDevice),hipMemcpyHostToDevice);
  printf("%d %d ....\n",numElectrons,NumColls);
  MonteTRun<<<blocks,threads>>>(DeviceParametersPointer);
  //Test<<<threads,blocks>>>(DeviceParametersPointer,DeviceParameters->RGAS);
  hipDeviceSynchronize();
  hipMemcpy(XOutput,DeviceParameters->XOutput,100000*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(YOutput,DeviceParameters->YOutput,100000*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(ZOutput,DeviceParameters->ZOutput,100000*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(TimeSumOutput,DeviceParameters->TimeSumOutput,100000*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(RGAS,DeviceParameters->RGAS,6*290*sizeof(double),hipMemcpyDeviceToHost);
  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(DeviceParametersPointer);
}
