#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#ifndef max
#define max(a,b) (((a) > (b)) ? (a) : (b))
 #endif
 #ifndef min
  #define min(a,b) (((a) < (b)) ? (a) : (b))
  #endif

#define maxx(x, y) (((x) > (y)) ? (x) : (y))
#define minn(x, y) (((x) < (y)) ? (x) : (y))
int main()
{
    return 0;
}
// cudamalloc functions
double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
double * LinearizeAndCopy(double** arr,int h,int w){

  double * pointer;

  double * temp = (double *)malloc(h*w*sizeof(double));

  for(int i=0;i<h;++i){

    for(int j = 0;j<w;++j){
      temp[i*w+j] =arr[i][j];
    }
  }

  hipMalloc((void **)&pointer,h*w*sizeof(double));
  hipMemcpy(pointer,temp,h*w*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
struct RM48Gen {
  double RVEC[1001],U[98];
  int IVEC = 0;
  int NVEC = 1000;
  int  I97, J97;
  double C;
  int IJKLIN = 54217137,KALLED;
  double NTOT2N = 0,NTOTIN =0 ,NTOT=-1, NTOT2=0;
};

// RM48 functions
__device__ double dmod(double x, double y) {
    return x - (int)(x/y) * y;
}



void SetupRM48Gens(struct RM48Gen* gen,int s,long long * seeds){
    for(int i=0;i<s;++i){
      gen[i].IJKLIN = seeds[i];
      gen[i].NTOTIN = 0;
      gen[i].NTOT2N = 0;
      gen[i].KALLED = 1;
      gen[i].NTOT = -1;
      gen[i].NVEC = 1000;
      gen[i].IVEC = 0;
    }
}


__device__ double MOD(double A,double B){
  return dmod(A,B);
}


__device__ void RM48(struct RM48Gen *RM48gen,double LENV){
  long MODCNS = 1000000000;
  double T,S,HALF,UNI;
  long long NTOT2N,I,J,K,L,M,NOW,IJ,KL;
  static double CD, CM, TWOM24,TWOM49 ,ONE, ZERO;
  static long long IJKL=0;
  int II,JJ,I24,LOOP2,IDUM;


  if(RM48gen->NTOT>=0) goto L50;
  IJKL = RM48gen->IJKLIN;
  RM48gen->NTOT = RM48gen->NTOTIN;
  RM48gen->NTOT2 = RM48gen->NTOT2N;

  IJ = IJKL/30082;
  KL = IJKL - 30082*IJ;
  I = MOD(IJ/177, 177) + 2;
  J = MOD(IJ, 177)     + 2;
  K = MOD(KL/169, 178) + 1;
  L = MOD(KL, 169);
  ONE = 1.;
  HALF = 0.5;
  ZERO = 0.;
  for( II= 1;II<= 97;++II){
  S = 0.;
  T = HALF;
  for(JJ= 1;JJ<= 48;++JJ){
    M = MOD(MOD(I*J,179)*K, 179);
    I = J;
    J = K;
    K = M;
    L = MOD(53*L+1, 169);
     if (MOD(L*M,64) >= 32)  S = S+T;
     T = HALF*T;
   }
 RM48gen->U[II] = S;
}
TWOM49 = T;
TWOM24 = ONE;
for(I24= 1;I24<= 24;++I24){
 TWOM24 = HALF*TWOM24;
}
RM48gen->C  =   362436.*TWOM24;
CD =  7654321.*TWOM24;
CM = 16777213.*TWOM24;
RM48gen->I97 = 97;
RM48gen->J97 = 33;

for(LOOP2 = 1;LOOP2<=RM48gen->NTOT2+1;++LOOP2){
  NOW = MODCNS;
  if (LOOP2 == RM48gen->NTOT2+1)  NOW=RM48gen->NTOT;
  if (NOW > 0)  {
      for(IDUM = 1;IDUM<= RM48gen->NTOT;++IDUM){
      UNI = RM48gen->U[RM48gen->I97]-RM48gen->U[RM48gen->J97];
      if (UNI < ZERO)  UNI=UNI+ONE;
      RM48gen->U[RM48gen->I97] = UNI;
      RM48gen->I97 = RM48gen->I97-1;
      if (RM48gen->I97== 0)  RM48gen->I97=97;
      RM48gen->J97 = RM48gen->J97-1;
      if (RM48gen->J97 == 0)  RM48gen->J97=97;
     RM48gen->C =RM48gen->C - CD;
      if (RM48gen->C < ZERO) RM48gen->C=RM48gen->C+CM;
      }
  }
}

  if (RM48gen->KALLED == 1) {
    RM48gen->KALLED = 0;
  return;

  }
  L50:
  for( RM48gen->IVEC= 1;RM48gen->IVEC<=LENV;RM48gen->IVEC+=1){
  UNI = RM48gen->U[RM48gen->I97]-RM48gen->U[RM48gen->J97];
  if (UNI < ZERO)  UNI=UNI+ONE;
  RM48gen->U[RM48gen->I97] = UNI;
  RM48gen->I97 = RM48gen->I97-1;
  if (RM48gen->I97 == 0)  RM48gen->I97=97;
  RM48gen->J97 = RM48gen->J97-1;
  if (RM48gen->J97== 0)  RM48gen->J97=97;
 RM48gen->C =RM48gen->C - CD;
  if (RM48gen->C < ZERO) RM48gen->C=RM48gen->C+CM;
  UNI = UNI-RM48gen->C;
  if (UNI < ZERO) UNI=UNI+ONE;
  RM48gen->RVEC[RM48gen->IVEC] = UNI;
//             Replace exact zeros by 2**-49
     if (UNI == ZERO){
        RM48gen->RVEC[RM48gen->IVEC] = TWOM49;
     }
  }
  RM48gen->NTOT = RM48gen->NTOT + LENV;
     if (RM48gen->NTOT >= MODCNS) {
     RM48gen->NTOT2 = RM48gen->NTOT2 + 1;
     RM48gen->NTOT = RM48gen->NTOT - MODCNS;
   }

   return;
}
__device__ double DRAND48(struct RM48Gen *RM48gen,double dummy){
  if (RM48gen->IVEC ==0 || RM48gen->IVEC>=RM48gen->NVEC){
    RM48(RM48gen,RM48gen->NVEC);
    RM48gen->IVEC = 1;
  }else{
    RM48gen->IVEC+=1;
  }
  return RM48gen->RVEC[RM48gen->IVEC];
}


__global__ void SetupRM48GensCuda(struct RM48Gen * gen){
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  /*gen[i].RVEC =(double*) malloc(1001*sizeof(double));
  gen[i].U =(double*) malloc(98*sizeof(double));
*/
  RM48(&(gen[i]),gen[i].NVEC);
}

/*__global__ void FreeRM48GensCuda(struct RM48Gen * gen){
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  free(gen[i].RVEC);
  free(gen[i].U);

}*/

__global__ extern void GetCollisions(double *ElectronEnergyStep, double* MaxCollisionFreqTotal,double* BP,double*  F1,
  double*  F2,double* Sqrt2M,double* TwoM,double* TwoPi,double* MaxCollisionFreq,double * VTMB,double * TimeSum,
  double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
  double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * T,double * AP,double * TotalCollisionFrequency,struct RM48Gen * gen){

  // function start
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  int MaxBoltzNumsUsed = 1;
  DRAND48(&(gen[i]),0.5);
  //R = curand_uniform( &state );
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = DRAND48(&(gen[i]),0.5);
    R2 = DRAND48(&(gen[i]),0.5);

    RNMX[j] = sqrt((-1*log((double)R1)))*cos((double)(R2*(*TwoPi)));
    RNMX[j+1] = sqrt(-1*log((double)R1))*sin((double)R2*(*TwoPi));

  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = DRAND48(&(gen[i]),0.5);
    T[i] = -1 * log(RandomNum)/(*MaxCollisionFreqTotal)+TDash;
    TDash = T[i];
    AP[i] = DirCosineZ1[i]*(*F2)*sqrt(EBefore[i]);
    EAfter = EBefore[i]+(AP[i]+(*BP)*T[i])*T[i];
    VelocityRatio = sqrt(EBefore[i]/EAfter);
    DCosineZ2 = DirCosineZ1[i] * VelocityRatio + T[i] * (*F2) / (2 * sqrt(EAfter));
    DCosineX2 = DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;

    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){
        R1 = DRAND48(&(gen[i]),0.5);
        R2 = DRAND48(&(gen[i]),0.5);
        RNMX[j] = sqrt((-1*log(R1)))*cos((R2*(*TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*(*TwoPi));
      }
      MaxBoltzNumsUsed = 1;
    }
    GasVelX[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelY[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelZ[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    VelocityX[i] = DCosineX2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityY[i] = DCosineY2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityZ[i] = DCosineZ2 * (*Sqrt2M) * sqrt(EAfter);
    COMEnergy[i] = (pow((VelocityX[i] - GasVelX[i]), 2) + pow((VelocityY[i] - GasVelY[i]), 2) + pow(
        (VelocityZ[i] - GasVelZ[i]),
        2)) / (*TwoM);
    iEnergyBins[i] = COMEnergy[i] / (*ElectronEnergyStep);
    iEnergyBins[i] = minn(iEnergyBins[i], 3999);
    RandomNum = DRAND48(&(gen[i]),0.5);

    TEST = TotalCollisionFrequency[(int)iEnergyBins[i]] / (*MaxCollisionFreq);
    if (RandomNum < TEST){
      TimeSum[i] +=T[i];
      return;
    }
  }
}



// function that will be called from the PyBoltz_Gpu class
extern "C" void MonteTGpu(double PElectronEnergyStep,double PMaxCollisionFreqTotal,double PEField, double PCONST1,double PCONST2,double PCONST3
, double Ppi,double PISIZE,double PNumMomCrossSectionPoints,double PMaxCollisionFreq, double * PVTMB, double PAngleFromZ, double PAngleFromX,
double PInitialElectronEnergy, double** PCollisionFrequency, double *PTotalCollisionFrequency, double ** PRGAS, double ** PEnergyLevels,
double ** PAngleCut,double ** PScatteringParameter, double * PINDEX, double * PIPN
){
  double * EIN = LinearizeAndCopy(PEnergyLevels,6,290);

  // Copying constants into device
  double * ElectronEnergyStep = SetupAndCopyDouble(&(PElectronEnergyStep),1);
  double * MaxCollisionFreqTotal = SetupAndCopyDouble(&(PMaxCollisionFreqTotal),1);
  double bp = PEField*PEField*PCONST1;
  double * BP = SetupAndCopyDouble(&(bp),1);
  double f1 = PEField*PCONST2;
  double * F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = PEField*PCONST3;
  double * F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = PCONST3*0.01;
  double * Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  double * TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = Ppi*2;
  double * TwoPi = SetupAndCopyDouble(&(twpi),1);
  double * ISize = SetupAndCopyDouble(&(PISIZE),1);
  double * NumPoints = SetupAndCopyDouble(&(PNumMomCrossSectionPoints),1);
  double * MaxCollisionFreq = SetupAndCopyDouble(&(PMaxCollisionFreq),1);

  //Copying arrays to device
  double * VTMB = SetupAndCopyDouble((PVTMB),6);
  double * X = SetupArrayOneVal(0,1000);
  double * Y = SetupArrayOneVal(0,1000);
  double * Z = SetupArrayOneVal(0,1000);
  double * TimeSum = SetupArrayOneVal(0,1000);
  double * DirCosineZ1 = SetupArrayOneVal(cos(PAngleFromZ),1000);
  double * DirCosineX1 = SetupArrayOneVal(sin(PAngleFromZ) * cos(PAngleFromX),1000);
  double * DirCosineY1 = SetupArrayOneVal(sin(PAngleFromZ) * sin(PAngleFromX),1000);
  double * EBefore = SetupArrayOneVal(PInitialElectronEnergy,1000);
  double * iEnergyBins = SetupArrayOneVal(0,1000);
  double * COMEnergy = SetupArrayOneVal(0,1000);
  double * VelocityX = SetupArrayOneVal(0,1000);
  double * VelocityY = SetupArrayOneVal(0,1000);
  double * VelocityZ = SetupArrayOneVal(0,1000);
  double * GasVelX = SetupArrayOneVal(0,1000);
  double * GasVelY = SetupArrayOneVal(0,1000);
  double * GasVelZ = SetupArrayOneVal(0,1000);
  double * T = SetupArrayOneVal(0,1000);
  double * AP = SetupArrayOneVal(0,1000);
  double * AngleFromZ = SetupArrayOneVal(PAngleFromZ,1000);
  double * CF = LinearizeAndCopy((double **)PCollisionFrequency,4000,290);
  double * ANGCT = LinearizeAndCopy((double **)PAngleCut,4000,290);
  double * SCA = LinearizeAndCopy((double **)PScatteringParameter,4000,290);
  double * INDEX = SetupAndCopyDouble((PINDEX),290);
  double * IPN = SetupAndCopyDouble((PIPN),290);
  double * RGAS = LinearizeAndCopy((double **)PRGAS,6,290);
  double * TotalCollisionFrequency = SetupAndCopyDouble(PTotalCollisionFrequency,4000);



  //RM48 stuff
  //struct RM48Gen* gen =(struct RM48Gen *)malloc(1000*sizeof(struct RM48Gen));
  long long * Seeds = (long long *)malloc(1000*sizeof(long long));
  for (int i=0;i<1000;i++){
    Seeds[i] = (i*54217137)%100000000;
  }

  //printf("%d\n",gen[0].IJKLIN);
  struct  RM48Gen* gens = (struct  RM48Gen*)malloc(1000*sizeof(struct  RM48Gen));
  SetupRM48Gens(gens,1000,Seeds);

  struct  RM48Gen* pointer;
  hipMalloc((void **)&pointer,1000*sizeof(struct RM48Gen));
  hipMemcpy(pointer,gens,1000*sizeof(struct  RM48Gen),hipMemcpyHostToDevice);
  double * TT = (double *)malloc(1000*sizeof(double));
  SetupRM48GensCuda<<<int(1000),1>>>(pointer);
  for(int i=0;i<1000;++i){
/*
    double ElectronEnergyStep, double MaxCollisionFreqTotal,double BP,double  F1,
      double  F2,double Sqrt2M,double TwoM,double TwoPi,double MaxCollisionFreq,double * VTMB,double * TimeSum,
      double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
      double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
      double * T,double * AP,double * TotalCollisionFrequency,struct RM48Gen * gen
*/
    GetCollisions<<<int(1000),1>>>(ElectronEnergyStep, MaxCollisionFreqTotal, BP,F1,
      F2,Sqrt2M,TwoM,TwoPi,MaxCollisionFreq, VTMB,TimeSum,
      DirCosineZ1, DirCosineX1, DirCosineY1, EBefore, iEnergyBins,
      COMEnergy, VelocityX, VelocityY,VelocityZ, GasVelX, GasVelY, GasVelZ,
      T, AP, TotalCollisionFrequency, pointer);
      printf("HERE\n");
      hipMemcpy(TT,T,1000*sizeof(double),hipMemcpyDeviceToHost);
      printf("%d %f\n",i,TT[0]);
  }
  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(pointer);
  hipFree(ElectronEnergyStep);
  hipFree(MaxCollisionFreqTotal);
  hipFree(BP);
  hipFree(F1);
  hipFree(F2);
  hipFree(Sqrt2M);
  hipFree(TwoM);
  hipFree(TwoPi);
  hipFree(ISize);
  hipFree(NumPoints);
  hipFree(MaxCollisionFreq);
  hipFree(VTMB);
  hipFree(X);
  hipFree(Y);
  hipFree(Z);
  hipFree(TimeSum);
  hipFree(DirCosineX1);
  hipFree(DirCosineY1);
  hipFree(DirCosineZ1);
  hipFree(EBefore);
  hipFree(iEnergyBins);
  hipFree(COMEnergy);
  hipFree(VelocityZ);
  hipFree(VelocityY);
  hipFree(VelocityX);
  hipFree(T);
  hipFree(GasVelX);
  hipFree(GasVelY);
  hipFree(GasVelZ);
  hipFree(AP);
  hipFree(AngleFromZ);
  hipFree(CF);
  hipFree(RGAS);
  hipFree(EIN);
  hipFree(ANGCT);
  hipFree(SCA);
  hipFree(INDEX);
  hipFree(IPN);
}
