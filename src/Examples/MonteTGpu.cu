#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define maxx(x, y) (((x) > (y)) ? (x) : (y))
#define minn(x, y) (((x) < (y)) ? (x) : (y))
int main()
{
    return 0;
}
// cudamalloc functions
double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
double * LinearizeAndCopy(double** arr,int h,int w){

  double * pointer;

  double * temp = (double *)malloc(h*w*sizeof(double));

  for(int i=0;i<h;++i){

    for(int j = 0;j<w;++j){
      temp[i*w+j] =arr[i][j];
    }
  }

  hipMalloc((void **)&pointer,h*w*sizeof(double));
  hipMemcpy(pointer,temp,h*w*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
struct RM48Gen {
  double RVEC[1001];
  int IVEC = 0;
  int NVEC = 1000;
  int  I97, J97;
  double U[98],C;
  int IJKLIN = 54217137,KALLED;
  double NTOT2N = 0,NTOTIN =0 ,NTOT=-1, NTOT2=0;
};

// RM48 functions
__device__ double dmod(double x, double y) {
    return x - (int)(x/y) * y;
}

__device__ double MOD(double A,double B){
  return dmod(A,B);
}


void SetupRM48Gens(struct RM48Gen* gen,int s,long long * seeds){
    for(int i=0;i<s;++i){
      gen[i].IJKLIN = seeds[i];
    }
}

__global__ extern void GetCollisions(double ElectronEnergyStep, double MaxCollisionFreqTotal,double BP,double  F1,
  double  F2,double Sqrt2M,double TwoM,double TwoPi,double MaxCollisionFreq,double * VTMB,double * TimeSum,
  double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
  double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * T,double * AP,double * TotalCollisionFrequency,long long * seed){

  // function start
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  int MaxBoltzNumsUsed = 1;
  //R = curand_uniform( &state );
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;
  for(int j=0;j<5;j+=2){
    R1 = 0;
    R2 = 0;
    RNMX[(int)j] = sqrt((double)(-1*log((double)R1)))*cos((double)(R2*TwoPi));
    RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*TwoPi);
  }
  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;
  while(1){
    RandomNum = 0;
    T[(int)i] = -1 * log((double)RandomNum)/MaxCollisionFreqTotal+TDash;
    TDash = T[i];
    AP[i] = DirCosineZ1[i]*F2*sqrt(EBefore[i]);
    EAfter = EBefore[i]+(AP[i]+BP*T[i])*T[i];
    VelocityRatio = sqrt(EBefore[i]/EAfter);
    DCosineZ2 = DirCosineZ1[i] * VelocityRatio + T[i] * F2 / (2 * sqrt(EAfter));
    DCosineX2 = DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;
    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){
        R1 = 0;
        R2 = 0;
        RNMX[j] = sqrt(-1*log(R1))*cos(R2*TwoPi);
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*TwoPi);
      }
      MaxBoltzNumsUsed = 1;
    }
    GasVelX[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelY[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelZ[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    VelocityX[i] = DCosineX2 * Sqrt2M * sqrt(EAfter);
    VelocityY[i] = DCosineY2 * Sqrt2M * sqrt(EAfter);
    VelocityZ[i] = DCosineZ2 * Sqrt2M * sqrt(EAfter);
    COMEnergy[i] = (pow((VelocityX[i] - GasVelX[i]), 2) + pow((VelocityY[i] - GasVelY[i]), 2) + pow(
        (VelocityZ[i] - GasVelZ[i]),
        2)) / TwoM;
    iEnergyBins[i] = (int)COMEnergy[i] / ElectronEnergyStep;
    iEnergyBins[i] = minn(iEnergyBins[i], 3999);
    RandomNum = 0;
    TEST = TotalCollisionFrequency[(int)iEnergyBins[i]] / MaxCollisionFreq;
    if (RandomNum < TEST){
      TimeSum[i] +=T[i];
      return;
    }
  }
}



// function that will be called from the PyBoltz_Gpu class
extern "C" void MonteTGpu(double PElectronEnergyStep,double PMaxCollisionFreqTotal,double PEField, double PCONST1,double PCONST2,double PCONST3
, double Ppi,double PISIZE,double PNumMomCrossSectionPoints,double PMaxCollisionFreq, double * PVTMB, double PAngleFromZ, double PAngleFromX,
double PInitialElectronEnergy, double** PCollisionFrequency, double *PTotalCollisionFrequency, double ** PRGAS, double ** PEnergyLevels,
double ** PAngleCut,double ** PScatteringParameter, double * PINDEX, double * PIPN
){
  double * EIN = LinearizeAndCopy(PEnergyLevels,6,290);

  // Copying constants into device
  double * ElectronEnergyStep = SetupAndCopyDouble(&(PElectronEnergyStep),1);
  double * MaxCollisionFreqTotal = SetupAndCopyDouble(&(PMaxCollisionFreqTotal),1);
  double bp = PEField*PEField*PCONST1;
  double * BP = SetupAndCopyDouble(&(bp),1);
  double f1 = PEField*PCONST2;
  double * F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = PEField*PCONST3;
  double * F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = PCONST3*0.01;
  double * Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  double * TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = Ppi*2;
  double * TwoPi = SetupAndCopyDouble(&(twpi),1);
  double * ISize = SetupAndCopyDouble(&(PISIZE),1);
  double * NumPoints = SetupAndCopyDouble(&(PNumMomCrossSectionPoints),1);
  double * MaxCollisionFreq = SetupAndCopyDouble(&(PMaxCollisionFreq),1);

  //Copying arrays to device
  double * VTMB = SetupAndCopyDouble((PVTMB),6);
  double * X = SetupArrayOneVal(0,1000);
  double * Y = SetupArrayOneVal(0,1000);
  double * Z = SetupArrayOneVal(0,1000);
  double * TimeSum = SetupArrayOneVal(0,1000);
  double * DirCosineZ1 = SetupArrayOneVal(cos(PAngleFromZ),1000);
  double * DirCosineX1 = SetupArrayOneVal(sin(PAngleFromZ) * cos(PAngleFromX),1000);
  double * DirCosineY1 = SetupArrayOneVal(sin(PAngleFromZ) * sin(PAngleFromX),1000);
  double * EBefore = SetupArrayOneVal(PInitialElectronEnergy,1000);
  double * iEnergyBins = SetupArrayOneVal(0,1000);
  double * COMEnergy = SetupArrayOneVal(0,1000);
  double * VelocityX = SetupArrayOneVal(0,1000);
  double * VelocityY = SetupArrayOneVal(0,1000);
  double * VelocityZ = SetupArrayOneVal(0,1000);
  double * GasVelX = SetupArrayOneVal(0,1000);
  double * GasVelY = SetupArrayOneVal(0,1000);
  double * GasVelZ = SetupArrayOneVal(0,1000);
  double * T = SetupArrayOneVal(0,1000);
  double * AP = SetupArrayOneVal(0,1000);
  double * AngleFromZ = SetupArrayOneVal(PAngleFromZ,1000);
  double * CF = LinearizeAndCopy((double **)PCollisionFrequency,4000,290);
  double * ANGCT = LinearizeAndCopy((double **)PAngleCut,4000,290);
  double * SCA = LinearizeAndCopy((double **)PScatteringParameter,4000,290);
  double * INDEX = SetupAndCopyDouble((PINDEX),290);
  double * IPN = SetupAndCopyDouble((PIPN),290);
  double * RGAS = LinearizeAndCopy((double **)PRGAS,6,290);
  double * TotalCollisionFrequency = SetupAndCopyDouble(PTotalCollisionFrequency,4000);



  //RM48 stuff
  //struct RM48Gen* gen =(struct RM48Gen *)malloc(1000*sizeof(struct RM48Gen));
  long long * Seeds = (long long *)malloc(1000*sizeof(long long));
  for (int i=0;i<1000;i++){
    Seeds[i] = (i*25348)%100000;
  }

  //printf("%d\n",gen[0].IJKLIN);
  printf("HERE\n");

  long long * pointer;
  hipMalloc((void **)&pointer,1000*sizeof(long long));
  hipMemcpy(pointer,Seeds,1000*sizeof(long long),hipMemcpyHostToDevice);
  double * TT = (double *)malloc(1000*sizeof(double));
  for(int i=0;i<1000;++i){
    GetCollisions<<<int(1000),1>>>(*ElectronEnergyStep, *MaxCollisionFreqTotal, *BP,*F1,
      *F2,*Sqrt2M,*TwoM,*TwoPi,*MaxCollisionFreq, VTMB,TimeSum,
      DirCosineZ1, DirCosineX1, DirCosineY1, EBefore, iEnergyBins,
      COMEnergy, VelocityX, VelocityY,VelocityZ, GasVelX, GasVelY, GasVelZ,
      T, AP, TotalCollisionFrequency, pointer);
      hipMemcpy(TT,T,1000*sizeof(double),hipMemcpyDeviceToHost);
      printf("%d %f\n",i,TT[0]);
  }


  hipFree(ElectronEnergyStep);
  hipFree(MaxCollisionFreqTotal);
  hipFree(BP);
  hipFree(F1);
  hipFree(F2);
  hipFree(Sqrt2M);
  hipFree(TwoM);
  hipFree(TwoPi);
  hipFree(ISize);
  hipFree(NumPoints);
  hipFree(MaxCollisionFreq);
  hipFree(VTMB);
  hipFree(X);
  hipFree(Y);
  hipFree(Z);
  hipFree(TimeSum);
  hipFree(DirCosineX1);
  hipFree(DirCosineY1);
  hipFree(DirCosineZ1);
  hipFree(EBefore);
  hipFree(iEnergyBins);
  hipFree(COMEnergy);
  hipFree(VelocityZ);
  hipFree(VelocityY);
  hipFree(VelocityX);
  hipFree(T);
  hipFree(GasVelX);
  hipFree(GasVelY);
  hipFree(GasVelZ);
  hipFree(AP);
  hipFree(AngleFromZ);
  hipFree(CF);
  hipFree(RGAS);
  hipFree(EIN);
  hipFree(ANGCT);
  hipFree(SCA);
  hipFree(INDEX);
  hipFree(IPN);
}
