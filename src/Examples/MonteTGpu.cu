#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#ifndef max
#define max(a,b) (((a) > (b)) ? (a) : (b))
 #endif
 #ifndef min
  #define min(a,b) (((a) < (b)) ? (a) : (b))
  #endif

#define maxx(x, y) (((x) > (y)) ? (x) : (y))
#define minn(x, y) (((x) < (y)) ? (x) : (y))
int main()
{
    return 0;
}
// cudamalloc functions
double * SetupAndCopyDouble(double * data,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  hipMemcpy(pointer,data,s*sizeof(double),hipMemcpyHostToDevice);
  return pointer;
}

double * SetupArrayOneVal(double val,int s){
  double * pointer;
  hipMalloc((void **)&pointer,s*sizeof(double));
  double * temp = (double *)malloc(s*sizeof(double));
  for(int i=0;i<s;++i){
    temp[i] = val;
  }
  hipMemcpy(pointer,temp,s*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
double * LinearizeAndCopy(double** arr,int h,int w){

  double * pointer;

  double * temp = (double *)malloc(h*w*sizeof(double));

  for(int i=0;i<h;++i){

    for(int j = 0;j<w;++j){
      temp[i*w+j] =arr[i][j];
    }
  }

  hipMalloc((void **)&pointer,h*w*sizeof(double));
  hipMemcpy(pointer,temp,h*w*sizeof(double),hipMemcpyHostToDevice);
  free(temp);
  return pointer;
}
struct RM48Gen {
  double RVEC[1001],U[98];
  int IVEC = 0;
  int NVEC = 1000;
  int  I97, J97;
  double C;
  int IJKLIN = 54217137,KALLED;
  double NTOT2N = 0,NTOTIN =0 ,NTOT=-1, NTOT2=0;
};

// RM48 functions
__device__ double dmod(double x, double y) {
    return x - (int)(x/y) * y;
}



void SetupRM48Gens(struct RM48Gen* gen,int s,long long * seeds){
    for(int i=0;i<s;++i){
      gen[i].IJKLIN = seeds[i];
      gen[i].NTOTIN = 0;
      gen[i].NTOT2N = 0;
      gen[i].KALLED = 1;
      gen[i].NTOT = -1;
      gen[i].NVEC = 1000;
      gen[i].IVEC = 0;
    }
}


__device__ double MOD(double A,double B){
  return dmod(A,B);
}


__device__ void RM48(struct RM48Gen *RM48gen,double LENV){
  long MODCNS = 1000000000;
  double T,S,HALF,UNI;
  long long NTOT2N,I,J,K,L,M,NOW,IJ,KL;
  static double CD, CM, TWOM24,TWOM49 ,ONE, ZERO;
  static long long IJKL=0;
  int II,JJ,I24,LOOP2,IDUM;


  if(RM48gen->NTOT>=0) goto L50;
  IJKL = RM48gen->IJKLIN;
  RM48gen->NTOT = RM48gen->NTOTIN;
  RM48gen->NTOT2 = RM48gen->NTOT2N;

  IJ = IJKL/30082;
  KL = IJKL - 30082*IJ;
  I = MOD(IJ/177, 177) + 2;
  J = MOD(IJ, 177)     + 2;
  K = MOD(KL/169, 178) + 1;
  L = MOD(KL, 169);
  ONE = 1.;
  HALF = 0.5;
  ZERO = 0.;
  for( II= 1;II<= 97;++II){
  S = 0.;
  T = HALF;
  for(JJ= 1;JJ<= 48;++JJ){
    M = MOD(MOD(I*J,179)*K, 179);
    I = J;
    J = K;
    K = M;
    L = MOD(53*L+1, 169);
     if (MOD(L*M,64) >= 32)  S = S+T;
     T = HALF*T;
   }
 RM48gen->U[II] = S;
}
TWOM49 = T;
TWOM24 = ONE;
for(I24= 1;I24<= 24;++I24){
 TWOM24 = HALF*TWOM24;
}
RM48gen->C  =   362436.*TWOM24;
CD =  7654321.*TWOM24;
CM = 16777213.*TWOM24;
RM48gen->I97 = 97;
RM48gen->J97 = 33;

for(LOOP2 = 1;LOOP2<=RM48gen->NTOT2+1;++LOOP2){
  NOW = MODCNS;
  if (LOOP2 == RM48gen->NTOT2+1)  NOW=RM48gen->NTOT;
  if (NOW > 0)  {
      for(IDUM = 1;IDUM<= RM48gen->NTOT;++IDUM){
      UNI = RM48gen->U[RM48gen->I97]-RM48gen->U[RM48gen->J97];
      if (UNI < ZERO)  UNI=UNI+ONE;
      RM48gen->U[RM48gen->I97] = UNI;
      RM48gen->I97 = RM48gen->I97-1;
      if (RM48gen->I97== 0)  RM48gen->I97=97;
      RM48gen->J97 = RM48gen->J97-1;
      if (RM48gen->J97 == 0)  RM48gen->J97=97;
     RM48gen->C =RM48gen->C - CD;
      if (RM48gen->C < ZERO) RM48gen->C=RM48gen->C+CM;
      }
  }
}

  if (RM48gen->KALLED == 1) {
    RM48gen->KALLED = 0;
  return;

  }
  L50:
  for( RM48gen->IVEC= 1;RM48gen->IVEC<=LENV;RM48gen->IVEC+=1){
  UNI = RM48gen->U[RM48gen->I97]-RM48gen->U[RM48gen->J97];
  if (UNI < ZERO)  UNI=UNI+ONE;
  RM48gen->U[RM48gen->I97] = UNI;
  RM48gen->I97 = RM48gen->I97-1;
  if (RM48gen->I97 == 0)  RM48gen->I97=97;
  RM48gen->J97 = RM48gen->J97-1;
  if (RM48gen->J97== 0)  RM48gen->J97=97;
 RM48gen->C =RM48gen->C - CD;
  if (RM48gen->C < ZERO) RM48gen->C=RM48gen->C+CM;
  UNI = UNI-RM48gen->C;
  if (UNI < ZERO) UNI=UNI+ONE;
  RM48gen->RVEC[RM48gen->IVEC] = UNI;
//             Replace exact zeros by 2**-49
     if (UNI == ZERO){
        RM48gen->RVEC[RM48gen->IVEC] = TWOM49;
     }
  }
  RM48gen->NTOT = RM48gen->NTOT + LENV;
     if (RM48gen->NTOT >= MODCNS) {
     RM48gen->NTOT2 = RM48gen->NTOT2 + 1;
     RM48gen->NTOT = RM48gen->NTOT - MODCNS;
   }

   return;
}
__device__ double DRAND48(struct RM48Gen *RM48gen,double dummy){
  if (RM48gen->IVEC ==0 || RM48gen->IVEC>=RM48gen->NVEC){
    RM48(RM48gen,RM48gen->NVEC);
    RM48gen->IVEC = 1;
  }else{
    RM48gen->IVEC+=1;
  }
  return RM48gen->RVEC[RM48gen->IVEC];
}


__global__ void SetupRM48GensCuda(struct RM48Gen * gen){
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  /*gen[i].RVEC =(double*) malloc(1001*sizeof(double));
  gen[i].U =(double*) malloc(98*sizeof(double));
*/
  RM48(&(gen[i]),gen[i].NVEC);
}

/*__global__ void FreeRM48GensCuda(struct RM48Gen * gen){
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  free(gen[i].RVEC);
  free(gen[i].U);

}*/
__device__ int MBSortT(double RandomNum,double iEnergyBin,double * CF,double ISIZE,double NumPoints){
  int ISTEP,INCR,I;
  ISTEP = ISIZE;
  INCR = 0;
  for(int K = 0;K<12;++K){
      I = INCR;
      if(ISTEP==2){
        if(I==0){
          return I;
        }
        return I - 1;
      }
      I = INCR + ISTEP/2;
      if (I<= NumPoints){
        if(CF[(int)iEnergyBin*290+I]<RandomNum){
          INCR +=ISTEP;
        }
      }
      ISTEP = ISTEP/2;
  }
  if(I==0){
    return I;
  }
  return I - 1;
}
__global__ extern void GetCollisions(double *ElectronEnergyStep, double* MaxCollisionFreqTotal,double* BP,double*  F1,
  double*  F2,double* Sqrt2M,double* TwoM,double* TwoPi,double* MaxCollisionFreq,double * VTMB,double * TimeSum,
  double * DirCosineZ1,double * DirCosineX1,double * DirCosineY1,double * EBefore,double * iEnergyBins,
  double * COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * T,double * AP,double * TotalCollisionFrequency,struct RM48Gen * gen){

  // function start
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  int MaxBoltzNumsUsed = 1;
  DRAND48(&(gen[i]),0.5);
  //R = curand_uniform( &state );
  double RNMX[6]={0,0,0,0,0,0};
  double TDash = 0.0,R1,R2,RandomNum,TEST;

  for(int j=0;j<5;j+=2){
    R1 = DRAND48(&(gen[i]),0.5);
    R2 = DRAND48(&(gen[i]),0.5);

    RNMX[j] = sqrt((-1*log((double)R1)))*cos((double)(R2*(*TwoPi)));
    RNMX[j+1] = sqrt(-1*log((double)R1))*sin((double)R2*(*TwoPi));

  }

  double EAfter = 0.0,VelocityRatio,DCosineZ2,DCosineX2,DCosineY2;

  while(1){
    RandomNum = DRAND48(&(gen[i]),0.5);
    T[i] = -1 * log(RandomNum)/(*MaxCollisionFreqTotal)+TDash;
    TDash = T[i];
    AP[i] = DirCosineZ1[i]*(*F2)*sqrt(EBefore[i]);
    EAfter = EBefore[i]+(AP[i]+(*BP)*T[i])*T[i];
    VelocityRatio = sqrt(EBefore[i]/EAfter);
    DCosineZ2 = DirCosineZ1[i] * VelocityRatio + T[i] * (*F2) / (2 * sqrt(EAfter));
    DCosineX2 = DirCosineX1[i] * VelocityRatio;
    DCosineY2 = DirCosineY1[i] * VelocityRatio;
    RandomNum = 0;
    MaxBoltzNumsUsed += 1;

    if(MaxBoltzNumsUsed>6){
      for(int j=0;j<5;j+=2){
        R1 = DRAND48(&(gen[i]),0.5);
        R2 = DRAND48(&(gen[i]),0.5);
        RNMX[j] = sqrt((-1*log(R1)))*cos((R2*(*TwoPi)));
        RNMX[j+1] = sqrt(-1*log(R1))*sin(R2*(*TwoPi));
      }
      MaxBoltzNumsUsed = 1;
    }
    GasVelX[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelY[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    MaxBoltzNumsUsed += 1;
    GasVelZ[i] = VTMB[0] * RNMX[MaxBoltzNumsUsed - 1];
    VelocityX[i] = DCosineX2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityY[i] = DCosineY2 * (*Sqrt2M) * sqrt(EAfter);
    VelocityZ[i] = DCosineZ2 * (*Sqrt2M) * sqrt(EAfter);
    COMEnergy[i] = (pow((VelocityX[i] - GasVelX[i]), 2) + pow((VelocityY[i] - GasVelY[i]), 2) + pow(
        (VelocityZ[i] - GasVelZ[i]),
        2)) / (*TwoM);
    iEnergyBins[i] = COMEnergy[i] / (*ElectronEnergyStep);
    iEnergyBins[i] = min(iEnergyBins[i], 3999);
    RandomNum = DRAND48(&(gen[i]),0.5);

    TEST = TotalCollisionFrequency[(int)iEnergyBins[i]] / (*MaxCollisionFreq);
    if (RandomNum < TEST){
      TimeSum[i] +=T[i];
      return;
    }
  }
}

__global__ void ProcessCollisions(double *COMEnergy,double * VelocityX,double * VelocityY,double * VelocityZ,double * GasVelX,double * GasVelY,double * GasVelZ,
  double * AP, double * X,double * Y,double * Z, double *DirCosineX1,double *DirCosineY1,double *DirCosineZ1,double * iEnergyBin,double * CF,double * RGAS,double * EnergyLevels,
  double * INDEX, double * ANGCT, double * SCA, double * IPN, double * AngleFromZ, double * TwoPi, double * EBefore, double * Sqrt2M,
  double * TwoM,double *T,double * BP,double * F1,double * ISIZE,double * NumPoints,struct RM48Gen * gen)
  {
    int i = threadIdx.x+blockDim.x*blockIdx.x;

    int I;
    double VelocityInCOM,DXCOM,DYCOM,DZCOM,T2,A,B,VelocityBefore,RandomNum;
    double S1,S2,EI,EXTRA,RandomNum2,CosTheta,EpsilonOkhr,Theta,Phi,SinPhi,CosPhi;
    double ARG1,D,U,Q,CosZAngle,SinZAngle,ARGZ,CONST12;
    double VXLab,VYLab,VZLab;
    VelocityInCOM  =  ((*Sqrt2M) * sqrt(COMEnergy[i]));

    DXCOM = (VelocityX[i] - GasVelX[i]) / VelocityInCOM;
    DYCOM = (VelocityY[i] - GasVelY[i]) / VelocityInCOM;
    DZCOM = (VelocityZ[i] - GasVelZ[i]) / VelocityInCOM;

    T2 = T[i]*T[i];
    A = AP[i]*T[i];
    B = (*BP) * T2;

    VelocityBefore = (*Sqrt2M) * sqrt(EBefore[i]);

    A = T[i] * VelocityBefore;
    X[i] += DirCosineX1[i] * A;
    Y[i] += DirCosineY1[i] * A;
    Z[i] += DirCosineZ1[i] * A + T2 * (*F1);
    RandomNum = DRAND48(&(gen[i]),0.5);

    I = MBSortT(RandomNum,iEnergyBin[i], CF,(*ISIZE),(*NumPoints));
    while(CF[(int)iEnergyBin[i]*290+I]<RandomNum) I+=1;


    S1 = RGAS[I];
    EI = EnergyLevels[I];

    if(IPN[I]>0){
      RandomNum = DRAND48(&(gen[i]),0.5);
      EXTRA = RandomNum * (COMEnergy[i]-EI);
      EI = EXTRA + EI;
    }

    if(COMEnergy[i]<EI){
      EI = COMEnergy[i]-0.0001;
    }

    S2 = (S1*S1)/(S1 - 1.0);
    RandomNum = DRAND48(&(gen[i]),0.5);

    if(INDEX[I] == 1){
      RandomNum2 = DRAND48(&(gen[i]),0.5);
      CosTheta = 1.0 *RandomNum*ANGCT[(int)iEnergyBin[i]*290 + I];
      if(RandomNum2>SCA[(int)iEnergyBin[i]*290 + I]){
        CosTheta = -1.0 * CosTheta;
      }
    }else if(INDEX[I]==2){
      EpsilonOkhr = SCA[(int)iEnergyBin[i]*290 + I];
      CosTheta = 1.0 - (2.0 * RandomNum * (1.0 - EpsilonOkhr) / (1.0 + EpsilonOkhr * (1.0 - 2.0 * RandomNum)));
    }else{
      CosTheta = 1.0 - 2.0*RandomNum;
    }

    Theta = acos(CosTheta);
    RandomNum = DRAND48(&(gen[i]),0.5);
    Phi = (*TwoPi) * RandomNum;
    SinPhi = sin(Phi);
    CosPhi = cos(Phi);


    ARG1 = max(1.0 - S1*EI/COMEnergy[i],1e-20);

    D = 1.0 - CosTheta * sqrt(ARG1);
    U = (S1 - 1)*(S1-1)/ARG1;

    EBefore[i] = max(COMEnergy[i] * (1.0 - EI / (S1 * COMEnergy[i]) - 2.0 * D / S2), 1e-20);

    Q = min(sqrt((COMEnergy[i] / EBefore[i]) * ARG1) / S1,1.0);

    AngleFromZ[i] = asin(Q * sin(Theta));
    CosZAngle = cos(AngleFromZ[i]);

    if(CosTheta<0 && CosTheta*CosTheta>U){
      CosZAngle = -1 * CosZAngle;
    }
    SinZAngle = sin(AngleFromZ[i]);
    DZCOM = min(DZCOM,1.0);
    ARGZ = sqrt(DXCOM*DXCOM + DYCOM*DYCOM);
    if (ARGZ ==0){
      DirCosineZ1[i] = CosZAngle;
      DirCosineX1[i] = CosPhi * SinZAngle;
      DirCosineY1[i] = SinPhi * SinZAngle;
    }else{
      DirCosineZ1[i] = DZCOM * CosZAngle + ARGZ * SinZAngle * SinPhi;
      DirCosineY1[i] = DYCOM * CosZAngle + (SinZAngle / ARGZ) * (DXCOM * CosPhi - DYCOM * DZCOM * SinPhi);
      DirCosineX1[i] = DXCOM * CosZAngle - (SinZAngle / ARGZ) * (DYCOM * CosPhi + DXCOM * DZCOM * SinPhi);
    }

    CONST12 = (*Sqrt2M) * sqrt(EBefore[i]);
    VXLab = DirCosineX1[i] * CONST12 + GasVelX[i];
    VYLab = DirCosineY1[i] * CONST12 + GasVelY[i];
    VZLab = DirCosineZ1[i] * CONST12 + GasVelZ[i];

    EBefore[i] = (VXLab * VXLab + VYLab * VYLab + VZLab * VZLab) / (*TwoM);
    VelocityInCOM = ((*Sqrt2M) * sqrt(EBefore[i]));
    DirCosineX1[i] = VXLab / VelocityInCOM;
    DirCosineY1[i] = VYLab / VelocityInCOM;
    DirCosineZ1[i] = VZLab / VelocityInCOM;
}

// function that will be called from the PyBoltz_Gpu class
extern "C" double* MonteTGpu(double PElectronEnergyStep,double PMaxCollisionFreqTotal,double PEField, double PCONST1,double PCONST2,double PCONST3
, double Ppi,double PISIZE,double PNumMomCrossSectionPoints,double PMaxCollisionFreq, double * PVTMB, double PAngleFromZ, double PAngleFromX,
double PInitialElectronEnergy, double** PCollisionFrequency, double *PTotalCollisionFrequency, double ** PRGAS, double ** PEnergyLevels,
double ** PAngleCut,double ** PScatteringParameter, double * PINDEX, double * PIPN,double * output
){
  double * EIN = LinearizeAndCopy(PEnergyLevels,6,290);

  // Copying constants into device
  double * ElectronEnergyStep = SetupAndCopyDouble(&(PElectronEnergyStep),1);
  double * MaxCollisionFreqTotal = SetupAndCopyDouble(&(PMaxCollisionFreqTotal),1);
  double bp = PEField*PEField*PCONST1;
  double * BP = SetupAndCopyDouble(&(bp),1);
  double f1 = PEField*PCONST2;
  double * F1 = SetupAndCopyDouble(&(f1),1);
  double f2 = PEField*PCONST3;
  double * F2 = SetupAndCopyDouble(&(f2),1);
  double sqrt2m = PCONST3*0.01;
  double * Sqrt2M = SetupAndCopyDouble(&(sqrt2m),1);
  double twom = sqrt2m*sqrt2m;
  double * TwoM = SetupAndCopyDouble(&(twom),1);
  double twpi = Ppi*2;
  double * TwoPi = SetupAndCopyDouble(&(twpi),1);
  double * ISize = SetupAndCopyDouble(&(PISIZE),1);
  double * NumPoints = SetupAndCopyDouble(&(PNumMomCrossSectionPoints),1);
  double * MaxCollisionFreq = SetupAndCopyDouble(&(PMaxCollisionFreq),1);

  //Copying arrays to device
  double * VTMB = SetupAndCopyDouble((PVTMB),6);
  double * X = SetupArrayOneVal(0,1000);
  double * Y = SetupArrayOneVal(0,1000);
  double * Z = SetupArrayOneVal(0,1000);
  double * TimeSum = SetupArrayOneVal(0,1000);
  double * DirCosineZ1 = SetupArrayOneVal(cos(PAngleFromZ),1000);
  double * DirCosineX1 = SetupArrayOneVal(sin(PAngleFromZ) * cos(PAngleFromX),1000);
  double * DirCosineY1 = SetupArrayOneVal(sin(PAngleFromZ) * sin(PAngleFromX),1000);
  double * EBefore = SetupArrayOneVal(PInitialElectronEnergy,1000);
  double * iEnergyBins = SetupArrayOneVal(0,1000);
  double * COMEnergy = SetupArrayOneVal(0,1000);
  double * VelocityX = SetupArrayOneVal(0,1000);
  double * VelocityY = SetupArrayOneVal(0,1000);
  double * VelocityZ = SetupArrayOneVal(0,1000);
  double * GasVelX = SetupArrayOneVal(0,1000);
  double * GasVelY = SetupArrayOneVal(0,1000);
  double * GasVelZ = SetupArrayOneVal(0,1000);
  double * T = SetupArrayOneVal(0,1000);
  double * AP = SetupArrayOneVal(0,1000);
  double * AngleFromZ = SetupArrayOneVal(PAngleFromZ,1000);
  double * CF = LinearizeAndCopy((double **)PCollisionFrequency,4000,290);
  double * ANGCT = LinearizeAndCopy((double **)PAngleCut,4000,290);
  double * SCA = LinearizeAndCopy((double **)PScatteringParameter,4000,290);
  double * INDEX = SetupAndCopyDouble((PINDEX),290);
  double * IPN = SetupAndCopyDouble((PIPN),290);
  double * RGAS = LinearizeAndCopy((double **)PRGAS,6,290);
  double * TotalCollisionFrequency = SetupAndCopyDouble(PTotalCollisionFrequency,4000);
  printf("%f\n",PCollisionFrequency[0][0] );


  //RM48 stuff
  //struct RM48Gen* gen =(struct RM48Gen *)malloc(1000*sizeof(struct RM48Gen));
  long long * Seeds = (long long *)malloc(1000*sizeof(long long));
  for (int i=0;i<1000;i++){
    Seeds[i] = (i*54217137)%100000000;
  }
  int f = 0;
  //printf("%d\n",gen[0].IJKLIN);
  struct  RM48Gen* gens = (struct  RM48Gen*)malloc(1000*sizeof(struct  RM48Gen));
  SetupRM48Gens(gens,1000,Seeds);

  struct  RM48Gen* pointer;
  hipMalloc((void **)&pointer,1000*sizeof(struct RM48Gen));
  hipMemcpy(pointer,gens,1000*sizeof(struct  RM48Gen),hipMemcpyHostToDevice);
  double * TT = (double *)malloc(1000*sizeof(double));
  SetupRM48GensCuda<<<int(1000),1>>>(pointer);
  char * str;
  for(int i=0;i<10000;++i){
    GetCollisions<<<int(1000),1>>>(ElectronEnergyStep, MaxCollisionFreqTotal, BP,F1,
      F2,Sqrt2M,TwoM,TwoPi,MaxCollisionFreq, VTMB,TimeSum,
      DirCosineZ1, DirCosineX1, DirCosineY1, EBefore, iEnergyBins,
      COMEnergy, VelocityX, VelocityY,VelocityZ, GasVelX, GasVelY, GasVelZ,
      T, AP, TotalCollisionFrequency, pointer);
      ProcessCollisions<<<int(1000),1>>>(COMEnergy,VelocityX,VelocityY, VelocityZ, GasVelX,GasVelY, GasVelZ,
    AP, X, Y, Z, DirCosineX1,DirCosineY1,DirCosineZ1,iEnergyBins, CF, RGAS,EIN,
      INDEX,ANGCT, SCA, IPN, AngleFromZ,  TwoPi,  EBefore, Sqrt2M, TwoM,T,BP,F1,ISize,NumPoints,pointer);

      if(((i)%(10000/100))==0){
        hipMemcpy(&output[0*100000+f*1000],X,1000*sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(&output[1*100000+f*1000],Y,1000*sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(&output[2*100000+f*1000],Z,1000*sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(&output[3*100000+f*1000],TimeSum,1000*sizeof(double),hipMemcpyDeviceToHost);
        f+=1;
      }
      if(i!=0&& double(int(log2(i)))==log2(i)){
        printf("%d analyzed collisions\n", i );
      }
  }
  printf("HERE\n");

  //FreeRM48GensCuda<<<int(1000),1>>>(pointer);
  hipFree(pointer);
  hipFree(ElectronEnergyStep);
  hipFree(MaxCollisionFreqTotal);
  hipFree(BP);
  hipFree(F1);
  hipFree(F2);
  hipFree(Sqrt2M);
  hipFree(TwoM);
  hipFree(TwoPi);
  hipFree(ISize);
  hipFree(NumPoints);
  hipFree(MaxCollisionFreq);
  hipFree(VTMB);
  hipFree(X);
  hipFree(Y);
  hipFree(Z);
  hipFree(TimeSum);
  hipFree(DirCosineX1);
  hipFree(DirCosineY1);
  hipFree(DirCosineZ1);
  hipFree(EBefore);
  hipFree(iEnergyBins);
  hipFree(COMEnergy);
  hipFree(VelocityZ);
  hipFree(VelocityY);
  hipFree(VelocityX);
  hipFree(T);
  hipFree(GasVelX);
  hipFree(GasVelY);
  hipFree(GasVelZ);
  hipFree(AP);
  hipFree(AngleFromZ);
  hipFree(CF);
  hipFree(RGAS);
  hipFree(EIN);
  hipFree(ANGCT);
  hipFree(SCA);
  hipFree(INDEX);
  hipFree(IPN);
  return output;
}
